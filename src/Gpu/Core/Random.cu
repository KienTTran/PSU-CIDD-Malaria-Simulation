#include "hip/hip_runtime.h"
//
// Created by kient on 6/17/2023.
//

#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include "Random.cuh"
#include "Gpu/Utils/Utils.cuh"
#include "Model.h"
#include "Core/Config/Config.h"

GPU::Random::Random() {
    d_states = nullptr;
}

GPU::Random::~Random() {
    hipFree(d_states);
}

__global__ void setup(int num,hiprandState *state, long seed)
{
    auto id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < num) hiprand_init(seed + id, 0, 0, &state[id]);
}

struct setupCurandStates
{
    long seed_;
    setupCurandStates(long seed) : seed_(seed) {}
    __device__
    hiprandState operator()(int id){
        hiprandState s;
        hiprand_init(seed_, id, 0, &s);
        return s;
    }
};

void GPU::Random::init_curand_states(ThrustTVectorDevice<hiprandState> &d_curand_states, int size, long seed){
    thrust::transform(thrust::counting_iterator<int>(0),
                      thrust::counting_iterator<int>(size),
                      d_curand_states.begin(),
                      setupCurandStates(seed));
}

void GPU::Random::init(int n, unsigned long seed, int n_threads, bool debug) {
    free();
    check_cuda_error(hipMalloc((void **) &d_states, sizeof(hiprandState) * n));
    int n_threads_ = (Model::CONFIG == nullptr || n_threads != -1) ? n_threads : Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (n + n_threads - 1) / n_threads_;
    if(n_threads > 0)
        LOG_IF(debug,INFO) << "GPU Random initializing " << n_threads_ << " threads with seed: " << seed;
    else
        LOG_IF(debug,INFO) << "GPU Random initializing default threads with seed: " << seed;
    setup<<<n_blocks,n_threads_>>>(n,d_states, seed);
    check_cuda_error(hipDeviceSynchronize());
    check_cuda_error(hipPeekAtLastError());
}

void GPU::Random::free() {
    hipFree(d_states);
}

/*
 * hiprand_uniform (hiprandState_t *state) - (0.0, 1.0]
 * Have to convert to [0.0,1.0) like GSL
 * */
__device__ double curand_uniform_gsl_double(double rand, double min, double max){
    return min + ((rand - 1.0e-6) * (max - min));
}

__device__ int curand_uniform_gsl_int(double rand, int min, int max){
    return int(min + ((rand - 1.0e-6) * (max - min)));
}

__device__ double curand_uniform_double_min_max(double rand,double min, double max){
    return min + (curand_uniform_gsl_double(rand,0.0,1.0) * (max - min));
}

__device__ int curand_uniform_int_min_max(double rand,int min, int max){
    return int(min + (curand_uniform_gsl_double(rand,0.0,1.0) * (max - min)));
}

__global__ void random_uniform_kernel_double(hiprandState *d_state, int n, double *d_randoms, double from, double to){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n; index += stride){
        d_randoms[index] = curand_uniform_double_min_max(hiprand_uniform_double(&local_state),from,to);
    }
    d_state[thread_index] = local_state;
}

__global__ void random_uniform_kernel_int(hiprandState *d_state, int n, int *d_randoms, int from, int to){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n; index += stride){
        d_randoms[index] = curand_uniform_int_min_max(hiprand_uniform_double(&local_state),from,to);
    }
    d_state[thread_index] = local_state;
}

/*
 * Random unifrom from [min,max) like GSL
 * */
ThrustTVectorDevice<double> GPU::Random::random_uniform_double_min_max(int size, double from, double to){
    ThrustTVectorDevice<double> d_randoms(size);
    int n_threads = Model::CONFIG == nullptr ? 1024 : Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (size + n_threads + 1) / n_threads;
    random_uniform_kernel_double<<<n_blocks, n_threads>>>(d_states, size, thrust::raw_pointer_cast(d_randoms.data()), from, to);
    return d_randoms;
}

ThrustTVectorDevice<int> GPU::Random::random_uniform_int_min_max(int size, int from, int to){
    ThrustTVectorDevice<int> d_randoms(size);
    int n_threads = Model::CONFIG == nullptr ? 1024 : Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (size + n_threads + 1) / n_threads;
    random_uniform_kernel_int<<<n_blocks, n_threads>>>(d_states, size, thrust::raw_pointer_cast(d_randoms.data()), from, to);
    return d_randoms;
}

/*
 * Binomial are from
 * https://stackoverflow.com/questions/23561551/a-efficient-binomial-random-number-generator-code-in-java/23574723#23574723
 * to get O(Np) when p is small instead of O(N) in curand_binomial_naive
 * Not need to use Rejection because N is not very large, as mentioned in
 * https://peterchng.com/blog/2020/10/23/building-binomial-and-multinomial-samplers-in-java/
 */
// Function to generate random numbers from a binomial distribution
__device__ unsigned int curand_binomial(hiprandState *state, double p, unsigned int N)
{
    double log_q = log(1.0 - p);
    int count = 0;
    double sum = 0;
    while(true) {
        sum += log(hiprand_uniform_double(state)) / (N - count);
        if(sum < log_q) {
            return count;
        }
        count++;
    }
}

/*
 * Binomial naive from
 * https://peterchng.com/blog/2020/10/23/building-binomial-and-multinomial-samplers-in-java/
 */
__device__ unsigned int curand_binomial_naive(hiprandState *state, double p, unsigned int N)
{
    unsigned int count = 0;
    if(N <= 0) return 0;
    for (unsigned int i = 0; i < N; ++i)
    {
        double rand_num = hiprand_uniform_double(state);
        if (rand_num < p)
        {
            count++;
        }
    }
    return count;
}

/*
 * Multinomial are from gsl_multinomial
 * This is parallel multinomial on n_locations, with K draws at each location.
 * The output must have size n_locations*K
 * N is the number of trials, p is the probability of each outcome, n is the output
 */
__global__ void multinomial_kernel(hiprandState *d_state, int n_locations, int K, int N[], double p[], unsigned int n[],
                                   double norm[], double sum_p[], unsigned int sum_n[])
{
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n_locations; index += stride){
        if(N[index] <= 0) {
            for (size_t i = 0; i < K; i++){
                n[index * K + i] = 0;
            }
            return;
        }
        for (size_t i = 0; i < K; i++)
        {
            norm[index] += p[index*K+i];
        }

        __syncthreads();

        // Calculate multinomial distribution
        for (size_t i = 0; i < K; i++)
        {
            if (p[index*K+i] > 0.0)
            {
                n[index*K+i] = curand_binomial_naive(&local_state, p[index*K+i] / (norm[index] - sum_p[index]), N[index] - sum_n[index]);
            }
            else
            {
                n[index*K+i] = 0;
            }

            sum_p[index] += p[index*K+i];
            sum_n[index] += n[index*K+i];
        }
        __syncthreads();
    }
    d_state[thread_index] = local_state;
}

/*
 * Multinomial are from gsl_multinomial
 * This is parallel multinomial on n_locations, with d_n_trials at each location.
 * The output must have size n_locations*n_samples_each_location
 */
void GPU::Random::random_multinomial(int n_locations, int n_samples_each_location,
                                     ThrustTVectorDevice<int> d_n_trials,
                                     ThrustTVectorDevice<double> d_distributions,
                                     ThrustTVectorDevice<unsigned int> &d_samples){
    int n_threads = Model::CONFIG == nullptr ? 1024 : Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (n_locations + n_threads + 1) / n_threads;
    ThrustTVectorDevice<double> d_norm(n_locations,0.0);
    ThrustTVectorDevice<double> d_sum_p(n_locations,0.0);
    ThrustTVectorDevice<unsigned int> d_sum_n(n_locations,0);
    multinomial_kernel<<<n_blocks, n_threads>>>(d_states,
                                                n_locations,
                                                n_samples_each_location,
                                                thrust::raw_pointer_cast(d_n_trials.data()),
                                                thrust::raw_pointer_cast(d_distributions.data()),
                                                thrust::raw_pointer_cast(d_samples.data()),
                                                thrust::raw_pointer_cast(d_norm.data()),
                                                thrust::raw_pointer_cast(d_sum_p.data()),
                                                thrust::raw_pointer_cast(d_sum_n.data()));
    check_cuda_error(hipDeviceSynchronize());
    check_cuda_error(hipPeekAtLastError());
    d_norm.clear();
    d_sum_p.clear();
    d_sum_n.clear();
    ThrustTVectorDevice<double>().swap(d_norm);
    ThrustTVectorDevice<double>().swap(d_sum_p);
    ThrustTVectorDevice<unsigned int>().swap(d_sum_n);
}

__global__ void multinomial_sampling_kernel(int n_locations,
                                            int n_distributions_each_location,
                                            int n_samples_each_location,
                                            unsigned int *d_hit_per_object,
                                            int *d_index,
                                            int *d_sample_index,
                                            int *d_all_objects_index){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations; index += stride){
        int hit_index_from = index * n_distributions_each_location;
        for (auto i = 0; i < n_distributions_each_location; i++) {
            for(int j = 0;  j < d_hit_per_object[hit_index_from+i]; j++){
                d_sample_index[index*n_samples_each_location+d_index[index]] = d_all_objects_index[i];
                d_index[index]++;
            }
        }
    }
}

/*
 * This is GPU version of Random::multinomial_sampling
 * d_n_samples size is n_locations
 * d_distribution_all_locations size is n_locations*n_distributions_each_location
 * all_objects size is n_locations*n_distributions_each_location
 * d_sum_distribution size is n_locations
 * return size is n_locations*n_samples_each_location
 * */
template
TVector<GPU::Person*> GPU::Random::multinomial_sampling<GPU::Person>(int, int,
                                                           ThrustTVectorDevice<double>,TVector<GPU::Person*>,
                                                           ThrustTVectorDevice<double>&,bool);
template <class T>
TVector<T*> GPU::Random::multinomial_sampling(int n_locations, int n_samples_each_location,
                                              ThrustTVectorDevice<double> d_distribution_all_locations,
                                              TVector<T*> all_objects,
                                              ThrustTVectorDevice<double> &d_sum_distribution_all_locations,
                                              bool is_shuffled){
    int n_distributions_each_location = d_distribution_all_locations.size() / n_locations;
    TVector<T*> samples(n_locations*n_samples_each_location, nullptr);
    double d_sum = thrust::reduce(thrust::device, d_sum_distribution_all_locations.begin(), d_sum_distribution_all_locations.end(), 0.0, thrust::plus<double>());
    if(d_sum == 0.0){
        return samples;
    }else if(d_sum == n_locations*(-1)){
        TVector<double> h_sum(n_locations);
        for(int i = 0; i < n_locations; i++){
            int index_from = i*n_distributions_each_location;
            int index_to = index_from + n_distributions_each_location;
            h_sum[i] = thrust::reduce(thrust::device,
                                      d_distribution_all_locations.begin() + index_from,
                                      d_distribution_all_locations.begin() + index_to,
                                      0.0, thrust::plus<double>());
//            printf("GPU h_sum[%d] = %f\n",i,h_sum[i]);
        }
        d_sum_distribution_all_locations = h_sum;
    }

    ThrustTVectorDevice<unsigned int> d_hit_per_object(n_locations*n_distributions_each_location);
    ThrustTVectorDevice<int> d_n_trials(n_locations, n_samples_each_location);
    random_multinomial(n_locations, n_distributions_each_location,d_n_trials,d_distribution_all_locations,d_hit_per_object);

    ThrustTVectorDevice<int> d_index(n_locations*n_samples_each_location,0);
    ThrustTVectorDevice<int> d_sample_index(n_locations*n_samples_each_location,0);
    ThrustTVectorDevice<int> d_all_objects_index(n_locations*n_distributions_each_location);
    thrust::sequence(thrust::device, d_all_objects_index.begin(), d_all_objects_index.end(), 0, 1);

    int n_threads = Model::CONFIG == nullptr ? 1024 : Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (n_locations + n_threads + 1) / n_threads;
    multinomial_sampling_kernel<<<n_blocks, n_threads>>>(n_locations,
                                                         n_distributions_each_location,
                                                         n_samples_each_location,
                                                         thrust::raw_pointer_cast(d_hit_per_object.data()),
                                                         thrust::raw_pointer_cast(d_index.data()),
                                                         thrust::raw_pointer_cast(d_sample_index.data()),
                                                         thrust::raw_pointer_cast(d_all_objects_index.data()));
//
//    thrust::copy(d_sample_index.begin(), d_sample_index.end(), std::ostream_iterator<int>(std::cout, "\n"));
//    printf("\n");

    if(is_shuffled){
        thrust::default_random_engine g;
        for(int i = 0; i < n_locations; i++){
            int index_from = i*n_samples_each_location;
            int index_to = index_from + n_samples_each_location;
//            printf("Multinomial location %d shuffle from %d to %d\n",i,index_from,index_to);
            thrust::shuffle(thrust::device,
                            d_sample_index.begin() + index_from,
                            d_sample_index.begin() + index_to,
                            g);
        }
    }

//    thrust::copy(d_sample_index.begin(), d_sample_index.end(), std::ostream_iterator<double>(std::cout, "\n"));
//    printf("\n");

    for(int i = 0; i < d_sample_index.size(); i++){
        samples[i] = all_objects[d_sample_index[i]];
    }

    return samples;
}

/*
 * hiprand_uniform return (0,1]
 * gsl_rng_uniform return [0,1)
 * */
__global__ void random_uniform_kernel(hiprandState *d_state,int n_locations, int n_samples_each_location,
                                      double *d_sum_distribution_all_locations,double *d_uniform_sampling){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n_locations*n_samples_each_location; index += stride){
        int location_index = index / n_samples_each_location;
//        printf("kernel uniform index %d location_index %d, sum %f\n",
//               index,location_index,d_sum_distribution_all_locations[location_index]);
        d_uniform_sampling[index] = hiprand_uniform_double(&local_state) * d_sum_distribution_all_locations[location_index];
    }
    d_state[thread_index] = local_state;
}

/*
 * d_n_samples size is n_locations
 * d_distribution_all_locations size is n_locations*n_distributions_each_location
 * d_sum_weight size is n_locations
 * d_all_objects_index size is n_locations*n_samples_each_location
 * d_uniform_sampling size is n_locations*n_samples_each_location
 * d_uniform_sampling_index size is n_locations
 * d_sample_index size is n_locations*n_samples_each_location
 * */
__global__ void roulette_sampling_kernel(int n_locations,
                                         int n_samples_each_location,
                                         int n_distributions_each_location,
                                         double *d_distribution_all_locations,
                                         double *d_sum_weight,
                                         int *d_all_objects_index,
                                         double *d_uniform_sampling,
                                         int *d_uniform_sampling_index,
                                         int *d_sample_index){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations; index += stride){
        for (auto i = 0; i < n_distributions_each_location; i++) {
            int pi = index * n_distributions_each_location + i;
            d_sum_weight[index] += d_distribution_all_locations[pi];
//            printf("kernel roulette location %d, n_distributions_each_location %d, pi %d, d_distribution_all_locations %f d_uniform_sampling[%d] = %f sum_weight %f\n",
//                   index, n_distributions_each_location, pi, d_distribution_all_locations[pi],
//                   index*n_samples_each_location+d_uniform_sampling_index[index],
//                   d_uniform_sampling[index*n_samples_each_location+d_uniform_sampling_index[index]],
//                   d_sum_weight[index]);
            while (d_uniform_sampling_index[index] < n_samples_each_location
            && d_uniform_sampling[index*n_samples_each_location+d_uniform_sampling_index[index]] < d_sum_weight[index]) {
//                printf("  while kernel roulette location %d, n_distributions_each_location %d, pi %d, d_distribution_all_locations %f sum_weight %f "
//                       "d_uniform_sampling_index %d d_uniform_sampling[%d] = %f d_sample_index %d d_all_objects_index %d\n",
//                       index, n_distributions_each_location, pi, d_distribution_all_locations[pi], d_sum_weight[index],
//                       d_uniform_sampling_index[index],
//                       index*n_samples_each_location+d_uniform_sampling_index[index],
//                       d_uniform_sampling[index*n_samples_each_location+d_uniform_sampling_index[index]],
//                       d_sample_index[index*n_samples_each_location+d_uniform_sampling_index[index]],
//                       d_all_objects_index[pi]);
                d_sample_index[index*n_samples_each_location+d_uniform_sampling_index[index]] = d_all_objects_index[pi];
                d_uniform_sampling_index[index]++;
            }
            if (d_uniform_sampling_index[index] == n_samples_each_location) {
                return;
            }
        }
        __syncthreads();
    }
}

/*
 * This is GPU version of Random::roulette_sampling
 * d_n_samples size is n_locations
 * d_distribution_all_locations size is n_locations*distribution_size_each_location
 * all_objects size is n_locations*n_samples_each_location
 * d_sum_distribution_all_locations size is n_locations
 * return size is n_locations*n_samples_each_location
 * */
template
TVector<GPU::Person*> GPU::Random::roulette_sampling<GPU::Person>(int, int,
                                                        ThrustTVectorDevice<double>,TVector<GPU::Person*>,
                                                        ThrustTVectorDevice<double>&,bool);
template <class T>
TVector<T*> GPU::Random::roulette_sampling(int n_locations, int n_samples_each_location,
                                           ThrustTVectorDevice<double> d_distribution_all_locations,
                                           TVector<T*> all_objects,
                                           ThrustTVectorDevice<double> &d_sum_distribution_all_locations,
                                           bool is_shuffled){
    int n_distributions_each_location = d_distribution_all_locations.size() / n_locations;
    TVector<T*> samples(n_locations*n_samples_each_location, nullptr);
    double d_sum = thrust::reduce(thrust::device, d_sum_distribution_all_locations.begin(), d_sum_distribution_all_locations.end(), 0.0, thrust::plus<double>());
    if(d_sum == 0.0){
        return samples;
    }else if(d_sum == n_locations*(-1.0)){
        TVector<double> h_sum(n_locations);
        for(int i = 0; i < n_locations; i++){
            int index_from = i*n_distributions_each_location;
            int index_to = index_from + n_distributions_each_location;
            h_sum[i] = thrust::reduce(thrust::device,
                                                d_distribution_all_locations.begin() + index_from,
                                                d_distribution_all_locations.begin() + index_to,
                                                0.0, thrust::plus<double>());
//            printf("GPU Roulette h_sum[%d] = %f\n",i,h_sum[i]);
        }
        d_sum_distribution_all_locations = h_sum;
    }

    ThrustTVectorDevice<double> d_uniform_sampling(n_locations*n_samples_each_location);
    int n_threads = Model::CONFIG == nullptr ? 1024 : Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (d_uniform_sampling.size() + n_threads + 1) / n_threads;
    random_uniform_kernel<<<n_blocks, n_threads>>>(d_states,
                                                   n_locations,
                                                   n_samples_each_location,
                                                   thrust::raw_pointer_cast(d_sum_distribution_all_locations.data()),
                                                   thrust::raw_pointer_cast(d_uniform_sampling.data()));
    check_cuda_error(hipDeviceSynchronize());
    check_cuda_error(hipPeekAtLastError());

//    thrust::copy(d_uniform_sampling.begin(), d_uniform_sampling.end(), std::ostream_iterator<double>(std::cout, "\n"));
//    printf("\n");

    for(int i = 0; i < n_locations; i++){
        int index_from = i*n_samples_each_location;
        int index_to = index_from + n_samples_each_location;
//        printf("Roulette location %d sort from %d to %d\n",i,index_from,index_to);
        thrust::sort(thrust::device,
                     d_uniform_sampling.begin() + index_from,
                     d_uniform_sampling.begin() + index_to);
    }
//    thrust::copy(d_uniform_sampling.begin(), d_uniform_sampling.end(), std::ostream_iterator<double>(std::cout, "\n"));
//    printf("\n");

    ThrustTVectorDevice<int> d_sample_index(n_locations*n_samples_each_location,0);
    ThrustTVectorDevice<double> d_sum_weight(n_locations,0.0);
    ThrustTVectorDevice<int> d_all_objects_index(n_locations*n_distributions_each_location,0);
    thrust::sequence(thrust::device, d_all_objects_index.begin(), d_all_objects_index.end(), 0, 1);
    ThrustTVectorDevice<int> d_uniform_sampling_index(n_locations,0);
    n_blocks = (n_locations + n_threads + 1) / n_threads;
    roulette_sampling_kernel<<<n_blocks, n_threads>>>(n_locations,
                                                      n_samples_each_location,
                                                      n_distributions_each_location,
                                                      thrust::raw_pointer_cast(d_distribution_all_locations.data()),
                                                      thrust::raw_pointer_cast(d_sum_weight.data()),
                                                      thrust::raw_pointer_cast(d_all_objects_index.data()),
                                                      thrust::raw_pointer_cast(d_uniform_sampling.data()),
                                                      thrust::raw_pointer_cast(d_uniform_sampling_index.data()),
                                                      thrust::raw_pointer_cast(d_sample_index.data()));
    check_cuda_error(hipDeviceSynchronize());
    check_cuda_error(hipPeekAtLastError());

//    thrust::copy(d_sample_index.begin(), d_sample_index.end(), std::ostream_iterator<int>(std::cout, "\n"));
//    printf("\n");

    if(is_shuffled){
        thrust::default_random_engine g;
        for(int i = 0; i < n_locations; i++){
            int index_from = i*n_samples_each_location;
            int index_to = index_from + n_samples_each_location;
//            printf("Roulette location %d shuffle from %d to %d\n",i,index_from,index_to);
            thrust::shuffle(thrust::device,
                            d_sample_index.begin() + index_from,
                            d_sample_index.begin() + index_to,
                            g);
        }
    }

//    thrust::copy(d_sample_index.begin(), d_sample_index.end(), std::ostream_iterator<double>(std::cout, "\n"));
//    printf("\n");

    for(int i = 0; i < d_sample_index.size(); i++){
        samples[i] = all_objects[d_sample_index[i]];
    }
    return samples;
}

/*
 * https://stackoverflow.com/questions/16663281/generating-random-numbers-from-various-distributions-in-cuda
 * */
__device__ double curand_gamma (hiprandState localState, const double a, const double b){
    /* assume a > 0 */
    if (a < 1){
        double u = hiprand_uniform_double(&localState);
        return curand_gamma (localState, 1.0 + a, b) * pow (u, 1.0 / a);
    }
    {
        double x, v, u;
        double d = a - 1.0 / 3.0;
        double c = (1.0 / 3.0) / sqrt (d);

        while (1){
            do{
                x = hiprand_normal_double(&localState);
                v = 1.0 + c * x;
            } while (v <= 0);

            v = v * v * v;
            u = hiprand_uniform_double(&localState);

            if (u < 1 - 0.0331 * x * x * x * x)
                break;

            if (log (u) < 0.5 * x * x + d * (1 - v + log (v)))
                break;
        }
        return b * d * v;
    }
}


/*
 * https://stackoverflow.com/questions/16663281/generating-random-numbers-from-various-distributions-in-cuda
 * */
__device__ double curand_beta (hiprandState localState, const double a, const double b){

}
