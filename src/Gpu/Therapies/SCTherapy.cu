#include "hip/hip_runtime.h"
/*
 * File:   Therapy.cu
 * Author: nguyentran
 *
 * Created on June 3, 2013, 7:50 PM
 */

#include "SCTherapy.cuh"

#include "Core/Config/Config.h"
#include "Model.h"

GPU::SCTherapy::SCTherapy() : Therapy(), dosing_day {}, artemisinin_id { -1 } {}

GPU::SCTherapy::~SCTherapy() = default;

void GPU::SCTherapy::add_drug(int drug_id) {
  Therapy::add_drug(drug_id);
  if (drug_id == 0) {
    artemisinin_id = drug_id;
  }
}

int GPU::SCTherapy::get_arteminsinin_id() const {
  return artemisinin_id;
}

int GPU::SCTherapy::get_max_dosing_day() const {
  auto result = std::max_element(dosing_day.begin(), dosing_day.end());
  return *result;
}

// int Therapy::get_therapy_duration(int dosing_day) {
//     int result = 0;
//
//     for (int i = 0; i < drug_ids_.size(); i++) {
//         DrugType* dt = Model::CONFIG->gpu_drug_db()->get(drug_ids_[i]);
//         if (!dt->is_artemisinin()) {
//             result = std::max<int>(dt->get_duration(dosing_day), result);
//         }
//     }
//     return result;
// }
