#include "hip/hip_runtime.h"
//
// Created by kient on 6/17/2023.
//

#include "Utils.cuh"
#include "Population/Person.h"

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>
#include <thrust/remove.h>
#include "Model.h"
#include "Core/Config/Config.h"

GPU::Utils::Utils() {
}

GPU::Utils::~Utils() {
}

void GPU::Utils::init(){
}

struct CheckKeyT2
{
    template<typename T,typename T2>
    __host__ __device__
    bool operator()(thrust::tuple<T,T2> t0, thrust::tuple<T,T2> t1)
    {
        if(thrust::get<0>(t0) == thrust::get<0>(t1)) return true;
        else return false;
    }
};

struct SumValueT2
{
    template<typename T,typename T2>
    __host__ __device__
    thrust::tuple<T, T2> operator()(thrust::tuple<T,T2> t0, thrust::tuple<T,T2> t1)
    {
        return thrust::make_tuple(thrust::get<0>(t1),thrust::get<1>(t0) + thrust::get<1>(t1));
    }
};

struct IsValueZeroT2
{
    template<typename T,typename T2>
    __host__ __device__
    bool operator()(thrust::tuple<T,T2> t)
    {
        return thrust::get<1>(t) == 0;
    }
};

/*
 * This function is used to sum the values of an array grouped by value in another array
 * Two arrays must have the same size and size parameter is the length of output array
 * Output is a tuple with 2 elements: key and sum value of that key
 * For example:
 * input_keys = [0, 1, 2, 1, 3, 0] - location
 * input_values = [1, 2, 3, 4, 5, 6] - age
 * output = [(0,7), (1,6), (2,3), (3,5)]
*/
template //https://stackoverflow.com/a/51606460/9187675 - to call template functions from other files
ThrustTuple2Vector<int,int> GPU::Utils::sum_value_by_1key<int,int>(TVector<int>, TVector<int>, int);
template
ThrustTuple2Vector<double,double> GPU::Utils::sum_value_by_1key<double,double>(TVector<double>, TVector<double>, int);
template
ThrustTuple2Vector<int,double> GPU::Utils::sum_value_by_1key<int,double>(TVector<int>, TVector<double>, int);
template
ThrustTuple2Vector<double,int> GPU::Utils::sum_value_by_1key<double,int>(TVector<double>, TVector<int>, int);
template<typename T,typename T2>
ThrustTuple2Vector<T,T2> GPU::Utils::sum_value_by_1key(TVector<T> input_keys, TVector<T2> input_values, int size){

    ThrustTVectorDevice<T> device_keys = input_keys;
    ThrustTVectorDevice<T2> device_values = input_values;

    thrust::sort_by_key(thrust::device, device_keys.begin(), device_keys.end(), device_values.begin(), thrust::less<T>());

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(device_keys.begin(), device_values.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(device_keys.end(), device_values.end()));

    ThrustTuple2VectorDevice<T,T2> device_output_values(device_values.size());

    auto result = thrust::reduce_by_key(thrust::device,
                                        begin,
                                        end,
                                        begin,
                                        thrust::make_discard_iterator(),
                                        device_output_values.begin(),
                                        CheckKeyT2(),
                                        SumValueT2());
    int output_length = result.second - device_output_values.begin();
    ThrustTuple2Vector<T,T2> host_output_values(output_length);
    thrust::copy(device_output_values.begin(), device_output_values.begin() + output_length, host_output_values.begin());
    thrust::remove_if(host_output_values.begin(), host_output_values.end(), IsValueZeroT2());
    return host_output_values;
}

template<typename T>
__global__ void fill_missing_indices(thrust::tuple<T,int>* device_output_values, T* output, int size){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < size; i += stride){
        output[thrust::get<0>(device_output_values[i])] = thrust::get<1>(device_output_values[i]);
    }
}

/*
 * This function is used to count occurrence of each key in an array
 * Input is key array with size parameter is the length of output array
 * For example:
 * input_keys = [0, 1, 2, 1, 3, 0]
 * size = 4
 * output = [2, 2, 1, 1]
 * Index which is not in the key array will be count as 0
*/
template
TVector<int> GPU::Utils::count_by_1key<int>(TVector<int>, int);
template<typename T>
TVector<T> GPU::Utils::count_by_1key(TVector<T> input_keys, int size){
    ThrustTVectorDevice<T> device_keys = input_keys;
    thrust::sort(thrust::device, device_keys.begin(), device_keys.end(), thrust::less<T>());
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(device_keys.begin(), device_keys.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(device_keys.end(), device_keys.end()));

    ThrustTuple2VectorDevice<T,int> device_output_temp(input_keys.size());

    auto result = thrust::reduce_by_key(thrust::device,
                                        begin,
                                        end,//https://stackoverflow.com/questions/34250322/count-reduction-using-thrust
                                        thrust::make_zip_iterator(thrust::make_tuple(device_keys.begin(),thrust::make_constant_iterator(1))),
                                        thrust::make_discard_iterator(),
                                        device_output_temp.begin(),
                                        CheckKeyT2(),
                                        SumValueT2());
    ThrustTVectorDevice<T> device_output_values(size,0);
    TVector<T> host_output_values(size,0);
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (size + n_threads - 1) / n_threads;
    fill_missing_indices<T><<<n_blocks,n_threads>>>(thrust::raw_pointer_cast(device_output_temp.data()),
                                                      thrust::raw_pointer_cast(device_output_values.data()),
                                                      size);
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
    thrust::copy(device_output_values.begin(), device_output_values.end(), host_output_values.begin());
    return host_output_values;
}

/*
 * Get 3 vectors of from_location, to_location and moving_level for removing zero from
 * d_circulation_indices and d_n_circulations_all_loc_ml
 * */
__global__ void extract_locations_and_moving_levels_kernel(int n_locations,
                                                    int n_moving_levels,
                                                    thrust::tuple<int,int> *d_circulation_indices,
                                                    unsigned int *d_n_circulations_all_loc_ml,
                                                    int* d_from_indices,
                                                    int* d_to_indices,
                                                    int* d_moving_levels){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_moving_levels; index += stride) {
        if(d_n_circulations_all_loc_ml[index] == 0) return;
        int circulate_index = index / n_moving_levels;
        d_from_indices[index] = thrust::get<0>(d_circulation_indices[circulate_index]);
        d_to_indices[index] = thrust::get<1>(d_circulation_indices[circulate_index]);
        d_moving_levels[index] = index % n_moving_levels;
    }
}
