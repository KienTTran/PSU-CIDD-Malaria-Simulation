#include "hip/hip_runtime.h"
//
// Created by kient on 6/17/2023.
//

#include "Utils.cuh"
#include "Population/Person.h"

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>
#include <thrust/remove.h>
#include "Model.h"
#include "Core/Config/Config.h"

GPU::Utils::Utils() {
}

GPU::Utils::~Utils() {
}

void GPU::Utils::init(){
}

struct CheckKeyT2
{
    template<typename T,typename T2>
    __host__ __device__
    bool operator()(thrust::tuple<T,T2> t0, thrust::tuple<T,T2> t1)
    {
        if(thrust::get<0>(t0) == thrust::get<0>(t1)) return true;
        else return false;
    }
};

struct SumValueT2
{
    template<typename T,typename T2>
    __host__ __device__
    thrust::tuple<T, T2> operator()(thrust::tuple<T,T2> t0, thrust::tuple<T,T2> t1)
    {
        return thrust::make_tuple(thrust::get<0>(t1),thrust::get<1>(t0) + thrust::get<1>(t1));
    }
};

struct IsValueZeroT2
{
    template<typename T,typename T2>
    __host__ __device__
    bool operator()(thrust::tuple<T,T2> t)
    {
        return thrust::get<1>(t) == 0;
    }
};

/*
 * This function is used to sum the values of an array grouped by value in another array
 * Two arrays must have the same size and size parameter is the length of output array
 * Output is a tuple with 2 elements: key and sum value of that key
 * For example:
 * input_keys = [0, 1, 2, 1, 3, 0] - location
 * input_values = [1, 2, 3, 4, 5, 6] - age
 * output = [(0,7), (1,6), (2,3), (3,5)]
*/
template //https://stackoverflow.com/a/51606460/9187675 - to call template functions from other files
ThrustT2TupleVectorHost<int,int> GPU::Utils::sum_value_by_1key<int,int>(ThrustTVectorHost<int>, ThrustTVectorHost<int>, int);
template
ThrustT2TupleVectorHost<double,double> GPU::Utils::sum_value_by_1key<double,double>(ThrustTVectorHost<double>, ThrustTVectorHost<double>, int);
template
ThrustT2TupleVectorHost<int,double> GPU::Utils::sum_value_by_1key<int,double>(ThrustTVectorHost<int>, ThrustTVectorHost<double>, int);
template
ThrustT2TupleVectorHost<double,int> GPU::Utils::sum_value_by_1key<double,int>(ThrustTVectorHost<double>, ThrustTVectorHost<int>, int);
template<typename T,typename T2>
ThrustT2TupleVectorHost<T,T2> GPU::Utils::sum_value_by_1key(ThrustTVectorHost<T> input_keys, ThrustTVectorHost<T2> input_values, int size){

    ThrustTVectorDevice<T> device_keys = input_keys;
    ThrustTVectorDevice<T2> device_values = input_values;

    thrust::sort_by_key(thrust::device, device_keys.begin(), device_keys.end(), device_values.begin(), thrust::less<T>());

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(device_keys.begin(), device_values.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(device_keys.end(), device_values.end()));

    ThrustT2TupleVectorDevice<T,T2> device_output_values(device_values.size());

    auto result = thrust::reduce_by_key(thrust::device,
                                        begin,
                                        end,
                                        begin,
                                        thrust::make_discard_iterator(),
                                        device_output_values.begin(),
                                        CheckKeyT2(),
                                        SumValueT2());
    int output_length = result.second - device_output_values.begin();
    ThrustT2TupleVectorHost<T,T2> host_output_values(output_length);
    thrust::copy(device_output_values.begin(), device_output_values.begin() + output_length, host_output_values.begin());
    thrust::remove_if(host_output_values.begin(), host_output_values.end(), IsValueZeroT2());
    return host_output_values;
}

template<typename T>
__global__ void fill_missing_indices(thrust::tuple<T,int>* device_output_values, T* output, int size){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < size; i += stride){
        output[thrust::get<0>(device_output_values[i])] = thrust::get<1>(device_output_values[i]);
    }
}

/*
 * This function is used to count occurrence of each key in an array
 * Input is key array with size parameter is the length of output array
 * For example:
 * input_keys = [0, 1, 2, 1, 3, 0]
 * size = 4
 * output = [2, 2, 1, 1]
 * Index which is not in the key array will be count as 0
*/
template
ThrustTVectorHost<int> GPU::Utils::count_by_1key<int>(ThrustTVectorHost<int>, int);
template<typename T>
ThrustTVectorHost<T> GPU::Utils::count_by_1key(ThrustTVectorHost<T> input_keys, int size){
    ThrustTVectorDevice<T> device_keys = input_keys;
    thrust::sort(thrust::device, device_keys.begin(), device_keys.end(), thrust::less<T>());
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(device_keys.begin(), device_keys.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(device_keys.end(), device_keys.end()));

    ThrustT2TupleVectorDevice<T,int> device_output_temp(input_keys.size());

    auto result = thrust::reduce_by_key(thrust::device,
                                        begin,
                                        end,//https://stackoverflow.com/questions/34250322/count-reduction-using-thrust
                                        thrust::make_zip_iterator(thrust::make_tuple(device_keys.begin(),thrust::make_constant_iterator(1))),
                                        thrust::make_discard_iterator(),
                                        device_output_temp.begin(),
                                        CheckKeyT2(),
                                        SumValueT2());
    ThrustTVectorDevice<T> device_output_values(size,0);
    ThrustTVectorHost<T> host_output_values(size,0);
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int n_blocks = (size + n_threads - 1) / n_threads;
    fill_missing_indices<T><<<n_blocks,n_threads>>>(thrust::raw_pointer_cast(device_output_temp.data()),
                                                      thrust::raw_pointer_cast(device_output_values.data()),
                                                      size);
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
    thrust::copy(device_output_values.begin(), device_output_values.end(), host_output_values.begin());
    return host_output_values;
}

/*
 * Get 3 vectors of from_location, to_location and moving_level for removing zero from
 * d_circulation_indices and d_n_circulations_all_loc_ml
 * */
__global__ void extract_locations_and_moving_levels_kernel(int n_locations,
                                                    int n_moving_levels,
                                                    thrust::tuple<int,int> *d_circulation_indices,
                                                    unsigned int *d_n_circulations_all_loc_ml,
                                                    int* d_from_indices,
                                                    int* d_to_indices,
                                                    int* d_moving_levels){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_moving_levels; index += stride) {
        if(d_n_circulations_all_loc_ml[index] == 0) return;
        int circulate_index = index / n_moving_levels;
        d_from_indices[index] = thrust::get<0>(d_circulation_indices[circulate_index]);
        d_to_indices[index] = thrust::get<1>(d_circulation_indices[circulate_index]);
        d_moving_levels[index] = index % n_moving_levels;
    }
}
