//
// Created by kient on 3/28/2022.
//

#include "ChangeInterruptedFeedingRateEvent.cuh"

GPU::ChangeInterruptedFeedingRateEvent::ChangeInterruptedFeedingRateEvent(const int &location, const double &ifr, const int &at_time)
    : location{location},
      ifr{ifr} {
  time = at_time;
}

void GPU::ChangeInterruptedFeedingRateEvent::execute() {
  Model::CONFIG->mosquito_config().interrupted_feeding_rate[location] = ifr;
  LOG(INFO) << date::year_month_day{scheduler->calendar_date} << " : Change interrupted feeding rate at location " << location << " to " << ifr;
}