#include "hip/hip_runtime.h"
/* 
 * File:   CirculateToTargetLocationNextDayEvent.cpp
 * Author: Merlin
 * 
 * Created on August 2, 2013, 10:57 AM
 */

#include "CirculateToTargetLocationNextDayEvent.cuh"
#include "Model.h"
#include "Core/Random.h"
#include "Core/Config/Config.h"
#include "ReturnToResidenceEvent.cuh"
#include "Gpu/Population/Person.cuh"


GPU::CirculateToTargetLocationNextDayEvent::CirculateToTargetLocationNextDayEvent() : target_location_(0) {}

GPU::CirculateToTargetLocationNextDayEvent::~CirculateToTargetLocationNextDayEvent() = default;

void GPU::CirculateToTargetLocationNextDayEvent::schedule_event(GPU::Scheduler *scheduler, GPU::Person *p, const int &target_location,
                                                           const int &time) {
  if (scheduler!=nullptr) {
    auto *e = new CirculateToTargetLocationNextDayEvent();
    e->dispatcher = p;
    e->set_target_location(target_location);
    e->time = time;

    p->add(e);
    scheduler->schedule_individual_event(e);
  }
}

std::string GPU::CirculateToTargetLocationNextDayEvent::name() {
  return "CirculateToTargetLocationNextDayEvent";
}

void GPU::CirculateToTargetLocationNextDayEvent::execute() {
  auto *person = dynamic_cast<GPU::Person *>(dispatcher);
  if(person->index() >= 1040 && person->index() <= 1045){
      printf("GPU::CirculateToTargetLocationNextDayEvent::execute() %d\n",person->index());
  }
  person->set_location(target_location_);

  if (target_location_!=person->residence_location()) {
    //if person already have return trip then no need to reschedule it
    //elase
    //Schedule for a return trip in next several days base on gamma distribution
    if (!person->has_return_to_residence_event()) {
      int length_of_trip = 0;
      while (length_of_trip < 1) {
        length_of_trip = static_cast<int>(std::round(
            Model::RANDOM->random_gamma(Model::CONFIG->circulation_info().length_of_stay_theta,
                                        Model::CONFIG->circulation_info().length_of_stay_k)));
      }

      //            std::cout << length_of_trip << std::endl;
      GPU::ReturnToResidenceEvent::schedule_event(Model::GPU_SCHEDULER, person,
                                             Model::GPU_SCHEDULER->current_time() + length_of_trip);

    }
  } else {
    //return by chance so we cancel all return event
    //cancel return trip and do nothing
    person->cancel_all_return_to_residence_events();
  }

}
