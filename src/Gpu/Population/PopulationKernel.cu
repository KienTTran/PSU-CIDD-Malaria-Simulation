#include "hip/hip_runtime.h"
//
// Created by kient on 12/31/2023.
//

#include <hiprand/hiprand_kernel.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include "PopulationKernel.cuh"
#include "Population.cuh"
#include "Model.h"
#include "Core/Config/Config.h"
#include "Gpu/MDC/ModelDataCollector.cuh"
#include "Spatial/SpatialModel.hxx"
#include "Core/Random.h"
#include "Gpu/Utils/Utils.cuh"
#include "Gpu/Core/Random.cuh"
#include "Gpu/Population/Properties/PersonIndexByLocationMovingLevel.cuh"
#include "Properties/PersonIndexGPU.cuh"
#include "ClonalParasitePopulation.cuh"
#include "SingleHostClonalParasitePopulations.cuh"
#include "ImmuneSystem.cuh"
#include <math.h>

GPU::PopulationKernel::PopulationKernel() {
}

void GPU::PopulationKernel::init() {
}
/*
 * Calculate number of circulations from each location
 * this if from poisson distribution with mean = popsize_residence_by_location * circulation_percent
 * d_n_circulations_from_locations is number of circulations from each location
 * */
__global__ void calculate_circulation_number(int n_locations,
                                             int* d_popsize_residence_by_location,
                                             double circulation_percent,
                                             hiprandState* d_state,
                                             int* d_n_circulations_from_locations) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n_locations; index += stride) {
        if (d_popsize_residence_by_location[index] == 0 || circulation_percent == 0.0) return;
        auto poisson_means = d_popsize_residence_by_location[index] * circulation_percent;
        d_n_circulations_from_locations[index] = hiprand_poisson(&d_state[index], poisson_means);
    }
    d_state[thread_index] = local_state;
}

/*
 * Calculate relative out movement to destination locations
 * this is get_v_relative_out_movement_to_destination in CPU v4.0 version
 * Output is relative out movement from each location to each destination location
 * So 9 locations will have 9*9 = 81 values
 * */
__global__ void calculate_circulation_probabilities(int n_locations,
                                                    int* d_popsize_residence_by_location,
                                                    double* d_spatial_model_parameters,
                                                    double* d_spatial_model_travels,
                                                    int* d_district_vector,
                                                    double* d_distance_vector,
                                                    double* d_relative_outmovement,
                                                    int* d_from_indices,
                                                    int* d_target_indices) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_locations; index += stride) {
        int from_location = index / n_locations;
        int target_location = index % n_locations;
        if(d_distance_vector[from_location * n_locations + target_location] == 0.0) return;
        double distance = d_distance_vector[from_location * n_locations + target_location];
        double kernel = pow(1 + (distance / d_spatial_model_parameters[2]), (-d_spatial_model_parameters[1]));
        double probability = pow(d_popsize_residence_by_location[from_location], d_spatial_model_parameters[0]) * kernel;
        probability = probability / (1 + d_spatial_model_travels[from_location] + d_spatial_model_travels[target_location] );
        if (d_district_vector[from_location] == static_cast<int>(d_spatial_model_parameters[3]) &&
            d_district_vector[target_location] == static_cast<int>(d_spatial_model_parameters[3])) {
            probability /= d_spatial_model_parameters[4];
        }
        d_relative_outmovement[index] = probability;
        d_from_indices[index] = from_location;
        d_target_indices[index] = target_location;
    }
}

void GPU::PopulationKernel::calculate_circulate_locations(int n_locations,ThrustTVectorDevice<int> &d_n_circulations_from_locations,ThrustTVectorDevice<double> &d_relative_outmovement_from_target_,
                                         ThrustTVectorDevice<int> &d_all_location_from_indices,ThrustTVectorDevice<int> &d_all_location_target_indices) {
    //Has to get pointer to device otherwise it will copy vector from host to device
    d_ce_popsize_residence_by_location = Model::GPU_DATA_COLLECTOR->popsize_residence_by_location();
    d_ce_spatial_model_parameters = Model::CONFIG->spatial_model()->getSpatialModelParameters();
    d_ce_spatial_model_travels = Model::CONFIG->spatial_model()->getSpatialModelTravels();
    d_ce_spatial_districts = Model::CONFIG->h_spatial_districts;
    d_ce_spatial_distances = Model::CONFIG->h_spatial_distances;

    //All probabilities because thrust run all arrays at the same time. If use 1 array then values are overwritten.
    d_n_circulations_from_locations.resize(n_locations,0);

    //Get circulations by location
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = ceil((d_n_circulations_from_locations.size() + n_threads - 1)/n_threads);
    calculate_circulation_number<<<block_size,n_threads>>>(n_locations,
                                                           thrust::raw_pointer_cast(d_ce_popsize_residence_by_location.data()),
                                                           Model::CONFIG->circulation_info().circulation_percent,
                                                           Model::GPU_RANDOM->d_states,
                                                           thrust::raw_pointer_cast(d_n_circulations_from_locations.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    //Get outmovement probabilities
    d_relative_outmovement_from_target_.resize(n_locations*n_locations,0.0);
    d_all_location_from_indices.resize(n_locations*n_locations,0);
    d_all_location_target_indices.resize(n_locations*n_locations,0);
    n_threads = Model::CONFIG->gpu_config().n_threads;
    block_size = (d_relative_outmovement_from_target_.size() + n_threads - 1)/n_threads;
    calculate_circulation_probabilities<<<block_size,n_threads>>>(Model::CONFIG->number_of_locations(),
                                                                  thrust::raw_pointer_cast(d_ce_popsize_residence_by_location.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_model_parameters.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_model_travels.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_districts.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_distances.data()),
                                                                  thrust::raw_pointer_cast(d_relative_outmovement_from_target_.data()),
                                                                  thrust::raw_pointer_cast(d_all_location_from_indices.data()),
                                                                  thrust::raw_pointer_cast(d_all_location_target_indices.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
}

/*
 * Calculate moving level density at each location, output size is n_location*n_moving_level
 * This value is for each destination locations
 * */
__global__ void calculate_moving_level_density_kernel(int n_locations,
                                               int n_moving_levels,
                                               thrust::tuple<int,int> *d_circulation_indices,
                                               int* d_popsize_by_moving_level,
                                               double* d_moving_level_value,
                                               double* d_moving_level_density) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_moving_levels; index += stride) {
        int circulate_loc_index = index / n_moving_levels;
        int from_location = thrust::get<0>(d_circulation_indices[circulate_loc_index]);
        int moving_level = index % n_moving_levels;
        if(d_popsize_by_moving_level[from_location*n_moving_levels+moving_level] == 0 || d_moving_level_value[moving_level] == 0.0) return;
        d_moving_level_density[index] = d_popsize_by_moving_level[from_location*n_moving_levels+moving_level] * d_moving_level_value[moving_level];
    }
}

void GPU::PopulationKernel::calculate_moving_level_density(ThrustTuple2VectorDevice<int,int> d_circulation_indices,ThrustTVectorDevice<double> &d_moving_level_density) {
    if(d_circulation_indices.size() == 0){
        return;
    }
    d_moving_level_density.resize(d_circulation_indices.size()*Model::CONFIG->circulation_info().number_of_moving_levels);
    d_ce_popsize_by_moving_level = Model::CONFIG->h_popsize_by_moving_level;
    d_ce_moving_level_value = Model::CONFIG->circulation_info().v_moving_level_value;
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = ceil((d_moving_level_density.size() + n_threads - 1)/n_threads);
    calculate_moving_level_density_kernel<<<block_size,n_threads>>>(d_circulation_indices.size(),
                                                             Model::CONFIG->circulation_info().number_of_moving_levels,
                                                             thrust::raw_pointer_cast(d_circulation_indices.data()),
                                                             thrust::raw_pointer_cast(d_ce_popsize_by_moving_level.data()),
                                                             thrust::raw_pointer_cast(d_ce_moving_level_value.data()),
                                                             thrust::raw_pointer_cast(d_moving_level_density.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
}

template <typename T>
struct notZero : public thrust::unary_function<T,bool> {
    __host__ __device__
    bool operator()(T x)
    {
        return x != 0;
    }
};

struct copyNotZero : public thrust::unary_function<unsigned int,bool>{
    __host__ __device__
    bool operator()(unsigned int x) {
        return x != 0;
    }
};

template <typename T>
struct isOne : public thrust::unary_function<T,bool> {
    __host__ __device__
    bool operator()(T x)
    {
        return x == 1;
    }
};

struct circulateLess{
    __host__ __device__
    bool operator()(const thrust::tuple<int,int,int,unsigned int>& t1, thrust::tuple<int,int,int,unsigned int>& t2)
    {
        if(t1.get<3>() < t2.get<3>())
            return true;
        if(t1.get<3>() > t2.get<3>())
            return false;
        return t1.get<3>() < t2.get<3>();
    }
};

/*
 * Get 3 vectors of from_location, target_location and moving_level for removing zero from
 * d_circulation_indices and d_n_circulations_all_loc_ml
 * */
__global__ void extract_locations_and_moving_levels(int n_locations,
                                          int n_moving_levels,
                                          thrust::tuple<int,int> *d_circulation_indices,
                                          unsigned int *d_n_circulations_all_loc_ml,
                                          int* d_from_indices,
                                          int* d_target_indices,
                                          int* d_moving_levels){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_moving_levels; index += stride) {
        if(d_n_circulations_all_loc_ml[index] == 0) return;
        int circulate_index = index / n_moving_levels;
        d_from_indices[index] = thrust::get<0>(d_circulation_indices[circulate_index]);
        d_target_indices[index] = thrust::get<1>(d_circulation_indices[circulate_index]);
        d_moving_levels[index] = index % n_moving_levels;
    }
}

/*
 * Combine 2 vectors to 1 vectors, for sorting
 * */
__global__ void zip_location_indices_and_n_circulations(int size,
                                          thrust::tuple<int,int,int>* d_circulations_all_loc_ml_indices_no_zero,
                                          unsigned int* d_n_circulations_all_loc_ml_no_zero,
                                          thrust::tuple<int,int,int,unsigned int>* d_circulate_all_loc_ml_today){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
        if(d_n_circulations_all_loc_ml_no_zero[index] == 0) return;
        d_circulate_all_loc_ml_today[index] = thrust::make_tuple(thrust::get<0>(d_circulations_all_loc_ml_indices_no_zero[index]),
                                                                     thrust::get<1>(d_circulations_all_loc_ml_indices_no_zero[index]),
                                                                     thrust::get<2>(d_circulations_all_loc_ml_indices_no_zero[index]),
                                                                     d_n_circulations_all_loc_ml_no_zero[index]);
    }
}

/*
 * Get random person index at each moving level in each location in all locations
 * Note that this is just filling 1 person index for each moving level in each location
 * so n_persons data in d_circulate_all_loc_ml_today is not used but will be embedded in d_circulate_person_indices_today
 * */
__global__ void fill_circulate_person_indices(int n_locations,
                                             int n_moving_levels,
                                             hiprandState *d_state,
                                             thrust::tuple<int,int,int,unsigned int> *d_circulate_all_loc_ml_today,
                                             int * d_popsize_by_loc_ml,
                                             thrust::tuple<int,int,int,unsigned int,int> *d_circulate_person_indices_today){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n_locations; index += stride) {
        if(thrust::get<3>(d_circulate_all_loc_ml_today[index]) == 0) return;
        int from_location = thrust::get<0>(d_circulate_all_loc_ml_today[index]);
        int target_location = thrust::get<1>(d_circulate_all_loc_ml_today[index]);
        int moving_level = thrust::get<2>(d_circulate_all_loc_ml_today[index]);
        unsigned int n_persons = thrust::get<3>(d_circulate_all_loc_ml_today[index]);
        int size = d_popsize_by_loc_ml[from_location*n_moving_levels+moving_level];
        if(size == 0 || n_persons == 0) return;
        /*
         * Random uniform to get person index
         * To get result same as gsl, using casting method, which is [0,n-1]
         * ceiling methos is [1,n]
         * https://github.com/nglee/so_answers/blob/master/cuda/170426/kernel.cu
         * */
//        printf("kernel %d from_location: %d, moving_level: %d, popsize_loc_ml: %d size: %d\n",index,
//               from_location,moving_level,popsize_loc_ml,d_n_circulations_all_loc_ml_no_zero[index]);
        if(n_persons == 1){
            int p_index = hiprand_uniform(&local_state) * size;
            d_circulate_person_indices_today[index] = thrust::make_tuple(from_location,target_location,moving_level,n_persons,p_index);
//        printf("kernel %d from_location: %d, moving_level: %d, popsize_loc_ml: %d, size: %d, p_index: %d\n",index,from_location,moving_level,
//               popsize_loc_ml,d_n_circulations_all_loc_ml_no_zero[index],p_index);
        }
        else{
            d_circulate_person_indices_today[index] = thrust::make_tuple(from_location,target_location,moving_level,n_persons,-1);
        }
    }
    d_state[thread_index] = local_state;
}

/*
 * To speed up circulation process using GPU, doing following steps:
 * 1. Calculate number of circulations and leavers, same as CPU
 * 2. Filter out location with zero circulation
 * 3. Parallel multinomial in all non-zero circulation locations,
 * each location select number of leavers in each moving level
 * 4. Sort the result of multinomial, so all location and moving level with 1 person will be filled first on GPU
 * 5. For the rest, fill on CPU
 * */
void GPU::PopulationKernel::perform_circulation_event() {
    auto tp_start = std::chrono::high_resolution_clock::now();

    /*
     * Calculate probability of leaving location in all locations (n_location*n_location)
     * Also get indices from and to arrays
     * */
    Model::GPU_POPULATION_KERNEL->calculate_circulate_locations(Model::CONFIG->number_of_locations(),
                                                         d_ce_n_circulations_from_locations,
                                                         d_ce_relative_outmovement_from_target,
                                                         d_ce_all_location_from_indices, d_ce_all_location_target_indices);

    /*
     * Calculate number of leavers in all locations
     * */
    ThrustTVectorDevice<unsigned int> d_num_leavers_from_target_(d_ce_relative_outmovement_from_target.size(), 0);
    Model::GPU_RANDOM->random_multinomial(Model::CONFIG->number_of_locations(),
                                          Model::CONFIG->number_of_locations(),
                                          d_ce_n_circulations_from_locations,
                                          d_ce_relative_outmovement_from_target,
                                          d_num_leavers_from_target_);
    size_t no_zero_size = thrust::count_if(d_num_leavers_from_target_.begin(), d_num_leavers_from_target_.end(), notZero<unsigned int>());
    /*
     * Remove zero values in d_num_leavers_from_target_ and d_n_circulations_from_locations
     * d_circulations_indices_no_zero index is not location index, its index is index of n_locations*n_locations
     * thrust::get<0>(d_circulations_indices_no_zero[i]) is from location index
     * thrust::get<1>(d_circulations_indices_no_zero[i]) is to location index
     * */
    ThrustTuple2VectorDevice<int,int> d_circulations_indices_no_zero(no_zero_size);

    /*
     * Remove zero values in d_num_leavers_from_target_ and d_n_circulations_from_locations
     * scan d_num_leavers_from_target_ and copy non-zero from & to locations to d_circulations_indices_no_zero
     * This is to reduce compute time
     */
    auto loc_index_begin = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.begin(), d_ce_all_location_target_indices.begin()));
    auto loc_index_end = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.end(), d_ce_all_location_target_indices.end()));
    thrust::copy_if(thrust::device,
                     loc_index_begin,
                     loc_index_end,
                     d_num_leavers_from_target_.begin(),
                     d_circulations_indices_no_zero.begin(),
                     copyNotZero());
    thrust::device_vector<unsigned int>::iterator nend = thrust::remove(thrust::device,d_num_leavers_from_target_.begin(),d_num_leavers_from_target_.end(),0);
    ThrustTVectorDevice<unsigned int> d_num_leavers_from_target_no_zero(no_zero_size);
    thrust::copy(d_num_leavers_from_target_.begin(),nend,d_num_leavers_from_target_no_zero.begin());

    /*
     * d_num_leavers_from_target_no_zero is non-zero leavers in all locations
     * */
    int total_leavers = thrust::reduce(thrust::device,d_num_leavers_from_target_no_zero.begin(),d_num_leavers_from_target_no_zero.end());
//    LOG_IF(total_leavers == 0, DEBUG) << "[Population] Update population circulation GPU total_leavers = 0";
    if(total_leavers == 0) return;

    /*
     * Calculate moving level density at each location, output size is n_location*n_moving_level
     * n_location is no-zero indices
     */
    ThrustTVectorDevice<double> d_moving_level_density;
    Model::GPU_POPULATION_KERNEL->calculate_moving_level_density(d_circulations_indices_no_zero,d_moving_level_density);

    ThrustTVectorDevice<unsigned int> d_n_circulations_all_loc_ml(d_moving_level_density.size(),0);
    Model::GPU_RANDOM->random_multinomial(d_circulations_indices_no_zero.size(),
                                         Model::CONFIG->circulation_info().number_of_moving_levels,
                                         d_num_leavers_from_target_no_zero,
                                         d_moving_level_density,
                                         d_n_circulations_all_loc_ml);
    /*
     * d_circulations_indices_no_zero and d_n_circulations_all_loc_ml not the same size
     */
//    TVector<unsigned int> h_n_circulations_all_loc_ml = d_n_circulations_all_loc_ml;
//    ThrustT2TupleVector<int,int> h_circulations_indices_no_zero = d_circulations_indices_no_zero;
//    for(int i = 0; i < h_n_circulations_all_loc_ml.size(); i++){
//        int circulate_loc_index = i / Model::CONFIG->circulation_info().number_of_moving_levels;
//        int from_location = thrust::get<0>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int target_location = thrust::get<1>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int moving_level = i % Model::CONFIG->circulation_info().number_of_moving_levels;
//        if(h_n_circulations_all_loc_ml[i] == 0) continue;
//        printf("%d from %d to %d moving level %d size %d\n",i,from_location,target_location,moving_level,
//               h_n_circulations_all_loc_ml[i]);
//    }
//    printf("\n");

    /*
     * Remove zero values in d_n_circulations_all_loc_ml
     * First, extract from and to locations from d_n_circulations_all_loc_ml to 2 vectors
     */
    d_ce_all_location_from_indices.resize(d_n_circulations_all_loc_ml.size());
    d_ce_all_location_target_indices.resize(d_n_circulations_all_loc_ml.size());
    d_ce_all_moving_levels.resize(d_n_circulations_all_loc_ml.size());
    if(d_n_circulations_all_loc_ml.size() == 0) return;
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = ceil((d_n_circulations_all_loc_ml.size() + n_threads - 1)/n_threads);
    extract_locations_and_moving_levels<<<block_size,n_threads>>>(d_circulations_indices_no_zero.size(),
                                                      Model::CONFIG->circulation_info().number_of_moving_levels,
                                                      thrust::raw_pointer_cast(d_circulations_indices_no_zero.data()),
                                                      thrust::raw_pointer_cast(d_n_circulations_all_loc_ml.data()),
                                                      thrust::raw_pointer_cast(d_ce_all_location_from_indices.data()),
                                                      thrust::raw_pointer_cast(d_ce_all_location_target_indices.data()),
                                                      thrust::raw_pointer_cast(d_ce_all_moving_levels.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
//    TVector<int> h_all_location_from_indices = d_all_location_from_indices;
//    TVector<int> h_all_location_target_indices = d_all_location_target_indices;
//    TVector<int> h_all_moving_levels = d_all_moving_levels;
//    for(int i = 0; i < h_n_circulations_all_loc_ml.size(); i++){
//        int circulate_loc_index = i / Model::CONFIG->circulation_info().number_of_moving_levels;
//        int from_location = thrust::get<0>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int target_location = thrust::get<1>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int moving_level = i % Model::CONFIG->circulation_info().number_of_moving_levels;
//        int from_location2 = h_all_location_from_indices[i];
//        int target_location2 = h_all_location_target_indices[i];
//        int moving_level2 = h_all_moving_levels[i];
//        if(h_n_circulations_all_loc_ml[i] == 0) continue;
//        printf("%d from %d to %d moving level %d (%d to %d moving level %d) size %d\n",i,from_location,target_location,moving_level,
//               from_location2,target_location2,moving_level2,h_n_circulations_all_loc_ml[i]);
//    }
//    printf("\n");
    /*
     * Remove zero values in d_n_circulations_all_loc_ml
     * scan d_n_circulations_all_loc_ml and copy non-zero from & to locations from 2 vectors
     * to d_circulations_all_loc_ml_indices_no_zero
     */
    no_zero_size = thrust::count_if(d_n_circulations_all_loc_ml.begin(), d_n_circulations_all_loc_ml.end(), notZero<unsigned int>());
    auto loc_index_begin_2 = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.begin(),
                                                                          d_ce_all_location_target_indices.begin(),
                                                                          d_ce_all_moving_levels.begin()));
    auto loc_index_end_2 = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.end(),
                                                                        d_ce_all_location_target_indices.end(),
                                                                        d_ce_all_moving_levels.end()));
    ThrustTuple3VectorDevice<int,int,int> d_circulations_all_loc_ml_indices_no_zero(no_zero_size);
    thrust::copy_if(thrust::device,
                    loc_index_begin_2,
                    loc_index_end_2,
                    d_n_circulations_all_loc_ml.begin(),
                    d_circulations_all_loc_ml_indices_no_zero.begin(),
                    copyNotZero());
    nend = thrust::remove(thrust::device,d_n_circulations_all_loc_ml.begin(),d_n_circulations_all_loc_ml.end(),0);
    ThrustTVectorDevice<unsigned int> d_n_circulations_all_loc_ml_no_zero(no_zero_size);
    thrust::copy(d_n_circulations_all_loc_ml.begin(),nend,d_n_circulations_all_loc_ml_no_zero.begin());

    /*
     * d_circulations_all_loc_ml_indices_no_zero is tuple of from_location, target_location and moving level
     * d_n_circulations_all_loc_ml_no_zero is non-zero circulation number in all locations and moving levels
     * d_n_circulations_all_loc_ml_no_zero and d_circulations_all_loc_ml_indices_no_zero are the same size and order
     * d_circulate_all_loc_ml_today is tuple of from_location, target_location, moving_level, person_index
     * */
    int total_circulations = thrust::reduce(thrust::device,d_n_circulations_all_loc_ml_no_zero.begin(),d_n_circulations_all_loc_ml_no_zero.end());
    ThrustTuple4VectorDevice<int,int,int,unsigned int> d_circulate_all_loc_ml_today(d_circulations_all_loc_ml_indices_no_zero.size(),thrust::make_tuple(-1,-1,-1,0));
    block_size = (d_circulate_all_loc_ml_today.size() + n_threads - 1)/n_threads;
    zip_location_indices_and_n_circulations<<<block_size,n_threads>>>(d_circulate_all_loc_ml_today.size(),
                                                                      thrust::raw_pointer_cast(d_circulations_all_loc_ml_indices_no_zero.data()),
                                                                      thrust::raw_pointer_cast(d_n_circulations_all_loc_ml_no_zero.data()),
                                                                      thrust::raw_pointer_cast(d_circulate_all_loc_ml_today.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    thrust::sort(thrust::device,d_circulate_all_loc_ml_today.begin(),d_circulate_all_loc_ml_today.end(),circulateLess());

//    ThrustT4TupleVector<int,int,int,unsigned int> h_circulate_all_loc_ml_today = d_circulate_all_loc_ml_today;
//    for(int i = 0; i < h_circulate_all_loc_ml_today.size(); i++){
//        int from_location = thrust::get<0>(h_circulate_all_loc_ml_today[i]);
//        int target_location = thrust::get<1>(h_circulate_all_loc_ml_today[i]);
//        int moving_level = thrust::get<2>(h_circulate_all_loc_ml_today[i]);
//        unsigned int size = thrust::get<3>(h_circulate_all_loc_ml_today[i]);
//        printf("%d from %d to %d moving level %d size %d\n",i,from_location,target_location,moving_level,size);
//    }
//    printf("\n");

    /*
     * Random persons based on d_n_circulations_all_loc_ml_no_zero
     * This needs to be done in 2 passes, 1st pass to fill all index with 1 person first, 1st pass is done on GPU
     * 2nd pass to fill all index with n_persons > 1, 2nd pass is done on CPU
     * In this is the first pass, fill all index in d_circulate_person_indices_today
     * with 1 person first
     * */
    ThrustTuple5VectorDevice<int,int,int,unsigned int,int> d_circulate_person_indices_today(total_circulations,thrust::make_tuple(-1,-1,-1,-1,-1));
    block_size = (d_n_circulations_all_loc_ml_no_zero.size() + n_threads - 1)/n_threads;
    fill_circulate_person_indices<<<block_size,n_threads>>>(d_circulate_all_loc_ml_today.size(),
                                                           Model::CONFIG->circulation_info().number_of_moving_levels,
                                                           Model::GPU_RANDOM->d_states,
                                                           thrust::raw_pointer_cast(d_circulate_all_loc_ml_today.data()),
                                                           thrust::raw_pointer_cast(d_ce_popsize_by_moving_level.data()),
                                                           thrust::raw_pointer_cast(d_circulate_person_indices_today.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    /*
     * Check if there is any index with n_persons > 1 and random on CPU
     * otherwise schedule person events
     * */
    ThrustTuple5VectorHost<int,int,int,unsigned int,int> h_circulate_person_indices_today = d_circulate_person_indices_today;
    auto *pi = Model::GPU_POPULATION->get_person_index<GPU::PersonIndexByLocationMovingLevel>();
    for(int i = 0; i < d_circulate_all_loc_ml_today.size();i++){
        int from_location = h_circulate_person_indices_today[i].get<0>();
        int target_location = h_circulate_person_indices_today[i].get<1>();
        int moving_level = h_circulate_person_indices_today[i].get<2>();
        int n_persons = h_circulate_person_indices_today[i].get<3>();
        auto size = static_cast<int>(pi->vPerson()[from_location][moving_level].size());
        if (size==0) continue;
        if(n_persons == 1){
            int p_index = h_circulate_person_indices_today[i].get<4>();
            GPU::Person* p = pi->vPerson()[from_location][moving_level][p_index];
            assert(p->host_state()!=GPU::Person::DEAD);
            p->today_target_locations()->push_back(target_location);
            p->randomly_choose_target_location();
//            printf("i %d GPU from %d to %d moving level %d n_persons %d p_index %d\n",
//                   i,
//                   from_location,target_location,moving_level,
//                   n_persons,p_index);
        }
        else{
            for(int j = 0; j < n_persons; j++) {
                int p_index = Model::RANDOM->random_uniform(size);
                GPU::Person* p = pi->vPerson()[from_location][moving_level][p_index];
                assert(p->host_state()!=GPU::Person::DEAD);
                p->today_target_locations()->push_back(target_location);
                p->randomly_choose_target_location();
//                printf("i %d j %d CPU from %d to %d moving level %d n_persons %d p_index %d\n",
//                       i,j,
//                       from_location,target_location,moving_level,
//                       n_persons,p_index);
            }
        }
    }

    if(Model::CONFIG->debug_config().enable_debug_text){
        auto lapse = std::chrono::high_resolution_clock::now() - tp_start;
        LOG_IF(Model::GPU_SCHEDULER->current_time() % Model::CONFIG->debug_config().log_interval == 0, INFO)
        << "[GPU Population] Update population circulation GPU (" << d_circulations_indices_no_zero.size() << " " << d_num_leavers_from_target_no_zero.size()
        << " " << total_leavers << " " << total_circulations << ") event time: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(lapse).count() << " ms ";
    }
}

void GPU::PopulationKernel::calculate_n_person_bitten_today(int n_locations,
                                                      ThrustTVectorDevice<double> &d_foi_all_locations,
                                                      ThrustTVectorDevice<int> &d_n_person_bitten_today_all_locations){


}

void GPU::PopulationKernel::perform_infection_event() {
    auto tp_start = std::chrono::high_resolution_clock::now();
    auto tracking_index = Model::GPU_SCHEDULER->current_time() % Model::CONFIG->number_of_tracking_days();

    /*
     * Calculate probability of leaving location in all locations (n_location*n_location)
     * Also get indices from and to arrays
     * */

    ThrustTVectorDevice<double> d_foi_all_locations;
    ThrustTVectorDevice<int> d_n_person_bitten_today_all_locations;
    calculate_n_person_bitten_today(Model::CONFIG->number_of_locations(),
                                    d_foi_all_locations,d_n_person_bitten_today_all_locations);
}

__global__ void update_current_foi_kernel(int size){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
    }
}

void GPU::PopulationKernel::update_current_foi(){

}

/*
 * Set update function inside kernel
 * To do this the base class of update function needs to be on GPU first
 * Remember to run
 * h_cpp->set_h_parasite_density_level(Model::CONFIG->parasite_density_level());
 * h_cpp->allocate_on_gpu();
 * before calling this function
 * */
__global__ void update_all_individuals_kernel2(int size,
                                              int current_time,
                                              int* latest_updated_time,
                                              ParasiteDensityLevel h_parasite_density_level,
                                              ImmuneSystemInformation *d_immune_system_information,
                                              int* d_update_function_type,
                                              double* d_last_update_log10_parasite_density,
                                              double* d_daily_fitness_multiple_infection,
                                              double* d_latest_immune_value,
                                              double* d_clonal_log10_parasite_density){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
        if (latest_updated_time[index] == current_time) return;
        int duration = current_time - latest_updated_time[index];
        if(d_update_function_type[index] == 1){
            d_clonal_log10_parasite_density[index] = h_parasite_density_level.log_parasite_density_asymptomatic;
        }
        if(d_update_function_type[index] == 2){
            double temp = d_immune_system_information->c_max*(1 - d_latest_immune_value[index])
                    + d_immune_system_information->c_min*d_latest_immune_value[index];
            d_clonal_log10_parasite_density[index] = d_last_update_log10_parasite_density[index]
                    + duration*(log10(temp) + log10(d_daily_fitness_multiple_infection[index]));
        }
    }
}


__global__ void all_individuals_update_parasite_kernel(int size,
                                               int current_time,
                                               ParasiteDensityLevel h_parasite_density_level,
                                               ImmuneSystemInformation *d_immune_system_information,
                                               GPU::PersonUpdateInfo* d_person_parasite_info){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
        /* Update parasites
         * This is equal to all_clonal_parasite_populations_->update();
         * */
        if (d_person_parasite_info[index].person_latest_update_time != current_time){
            for(int p_index = 0; p_index < d_person_parasite_info[index].parasite_size; p_index++){
//                if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                    printf("%d GPU all_individuals_update_parasite_kernel before update parasite %d %d %d %d %s %f\n",
//                           index,
//                           p_index,
//                           d_person_parasite_info[index].person_latest_update_time,
//                           current_time,
//                           d_person_parasite_info[index].parasite_update_function_type[p_index],
//                           d_person_parasite_info[index].parasite_genotype[p_index],
//                           d_person_parasite_info[index].parasite_last_update_log10_parasite_density[p_index]);
//                }
                int duration = current_time - d_person_parasite_info[index].person_latest_update_time;
                if(d_person_parasite_info[index].parasite_update_function_type[p_index] == 1){
                    d_person_parasite_info[index].parasite_last_update_log10_parasite_density[p_index] = h_parasite_density_level.log_parasite_density_asymptomatic;
                }
                if(d_person_parasite_info[index].parasite_update_function_type[p_index] == 2){
                    double temp = d_immune_system_information->c_max*(1 - d_person_parasite_info[index].person_latest_immune_value)
                                  + d_immune_system_information->c_min*d_person_parasite_info[index].person_latest_immune_value;
                    d_person_parasite_info[index].parasite_last_update_log10_parasite_density[p_index] =
                            d_person_parasite_info[index].parasite_last_update_log10_parasite_density[p_index]
                            + duration*(log10(temp) + log10(d_person_parasite_info[index].parasite_genotype_fitness_multiple_infection[p_index]));
                }
                d_person_parasite_info[index].person_latest_update_time = current_time;
//                if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                    printf("%d GPU all_individuals_update_parasite_kernel after update parasite %d %d %d %d %s %f\n",
//                           index,
//                           p_index,
//                           d_person_parasite_info[index].person_latest_update_time,
//                           current_time,
//                           d_person_parasite_info[index].parasite_update_function_type[p_index],
//                           d_person_parasite_info[index].parasite_genotype[p_index],
//                           d_person_parasite_info[index].parasite_last_update_log10_parasite_density[p_index]);
//                }
            }
        }
    }
}

__global__ void all_individuals_update_drug_kernel(int size,
                                                       int current_time,
                                                       ParasiteDensityLevel h_parasite_density_level,
                                                       ImmuneSystemInformation *d_immune_system_information,
                                                       GPU::PersonUpdateInfo* d_person_parasite_info){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
//        if(d_person_parasite_info[index].person_index == 1000){
//            printf("%d GPU update_all_individuals_kernel current_time %d\n",index,current_time);
//        }
        /* Update drug in blood
         * This is equal to drugs_in_blood_->update();
         * */
        for(int d_index = 0; d_index < d_person_parasite_info[index].drug_in_blood_size; d_index++){
            const int d_type_id = d_person_parasite_info[index].drug_in_blood_type_id[d_index];
            if(d_type_id != -1) {
//                if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                    printf("%d GPU update_all_individuals_kernel before update drug %d %d %d %f %f\n",
//                           index,
//                           d_person_parasite_info[index].drug_start_time[d_type_id],
//                           d_person_parasite_info[index].drug_last_update_time[d_type_id],
//                           d_type_id,
//                           d_person_parasite_info[index].drug_starting_value[d_type_id],
//                           d_person_parasite_info[index].drug_last_update_value[d_type_id]);
//                }
                d_person_parasite_info[index].drug_last_update_time[d_type_id] = current_time;
                const auto days = current_time - d_person_parasite_info[index].drug_start_time[d_type_id];
                if (days == 0) {
                    d_person_parasite_info[index].drug_last_update_value[d_type_id] = 0;
//                    if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                        printf("%d GPU update_all_individuals_kernel after update drug %d %d %d %f %f [1]\n",
//                               index,
//                               d_person_parasite_info[index].drug_start_time[d_type_id],
//                               d_person_parasite_info[index].drug_last_update_time[d_type_id],
//                               d_type_id,
//                               d_person_parasite_info[index].drug_starting_value[d_type_id],
//                               d_person_parasite_info[index].drug_last_update_value[d_type_id]);
//                    }
                    continue;
                }
                if (days <= d_person_parasite_info[index].drug_dosing_days[d_type_id]) {
                    if (d_type_id == 0) {
                        // drug is artemisinin
                        d_person_parasite_info[index].drug_last_update_value[d_type_id] =
                                d_person_parasite_info[index].drug_starting_value[d_type_id] + d_person_parasite_info[index].drug_rand_uniform_1[d_type_id];
//                        if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                            printf("%d GPU update_all_individuals_kernel after update drug %d %d %d %f %f [2]\n",
//                                   index,
//                                   d_person_parasite_info[index].drug_start_time[d_type_id],
//                                   d_person_parasite_info[index].drug_last_update_time[d_type_id],
//                                   d_type_id,
//                                   d_person_parasite_info[index].drug_starting_value[d_type_id],
//                                   d_person_parasite_info[index].drug_last_update_value[d_type_id]);
//                        }
                        continue;
                    }
                    d_person_parasite_info[index].drug_starting_value[d_type_id] += days >= 1 ? d_person_parasite_info[index].drug_rand_uniform_2[d_type_id] : 0;
                    d_person_parasite_info[index].drug_last_update_value[d_type_id] = d_person_parasite_info[index].drug_starting_value[d_type_id];
//                        if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                            printf("%d GPU update_all_individuals_kernel after update drug %d %d %d %f %f [3]\n",
//                                   index,
//                                   d_person_parasite_info[index].drug_start_time[d_type_id],
//                                   d_person_parasite_info[index].drug_last_update_time[d_type_id],
//                                   d_type_id,
//                                   d_person_parasite_info[index].drug_starting_value[d_type_id],
//                                   d_person_parasite_info[index].drug_last_update_value[d_type_id]);
//                        }
                    continue;
                } else {
                    const auto temp = fabs(d_person_parasite_info[index].drug_half_life[d_type_id] - 0.0) < d_person_parasite_info[index].drug_epsilon
                                      ? -100
                                      : -(days - d_person_parasite_info[index].drug_dosing_days[d_type_id]) * logf(2)
                                        / d_person_parasite_info[index].drug_half_life[d_type_id];  //-ai*t = - t* ln2 / tstar
                    if (exp(temp) <= (10.0 / 100.0)) {
                        d_person_parasite_info[index].drug_last_update_value[d_type_id] = 0;
//                    if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                        printf("%d GPU update_all_individuals_kernel after update drug %d %d %d %f %f [4]\n",
//                               index,
//                               d_person_parasite_info[index].drug_start_time[d_type_id],
//                               d_person_parasite_info[index].drug_last_update_time[d_type_id],
//                               d_type_id,
//                               d_person_parasite_info[index].drug_starting_value[d_type_id],
//                               d_person_parasite_info[index].drug_last_update_value[d_type_id]);
                        continue;
                    }
                    d_person_parasite_info[index].drug_last_update_value[d_type_id] = d_person_parasite_info[index].drug_starting_value[d_type_id] * exp(temp);
//                    if(d_person_parasite_info[index].person_index >= 1040 && d_person_parasite_info[index].person_index <= 1045){
//                        printf("%d GPU update_all_individuals_kernel after update drug %d %d %d %f %f [5]\n",
//                               index,
//                               d_person_parasite_info[index].drug_start_time[d_type_id],
//                               d_person_parasite_info[index].drug_last_update_time[d_type_id],
//                               d_type_id,
//                               d_person_parasite_info[index].drug_starting_value[d_type_id],
//                               d_person_parasite_info[index].drug_last_update_value[d_type_id]);
//                    }
                    continue;
                }
            }
        }
    }
}

__global__ void all_individuals_update_parasite_by_drug_kernel(int size,
                                                   int current_time,
                                                   ParasiteDensityLevel h_parasite_density_level,
                                                   ImmuneSystemInformation *d_immune_system_information,
                                                   GPU::PersonUpdateInfo* d_person_parasite_info){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
//        if(d_person_parasite_info[index].person_index == 1000){
//            printf("%d GPU update_all_individuals_kernel current_time %d\n",index,current_time);
//        }
        /* Update drug in blood
         * This is equal to all_clonal_parasite_populations_->update_by_drugs(drugs_in_blood_);
         * */
        for(int d_index = 0; d_index < d_person_parasite_info[index].drug_in_blood_size; d_index++){

        }
    }
}

struct person_has_parasites{
    __host__ __device__
    bool operator()(GPU::PersonUpdateInfo x){
        return x.parasite_size > 0;
    }
};

void GPU::PopulationKernel::update_all_individuals(){
    auto *pi = Model::GPU_POPULATION->get_person_index<GPU::PersonIndexGPU>();

    ThrustTVectorDevice<GPU::PersonUpdateInfo> d_person_update_info = pi->h_person_update_info();

//    for(int index = 1040; index <= 1045; index++){
//        if(pi->h_person_update_info()[index].parasite_size > 0){
//            for(int p_index = 0; p_index < pi->h_person_update_info()[index].parasite_size; p_index++){
//                printf("%d %d HOST update_all_individuals before before update parasite %d %d %d %d %s %f\n",
//                       Model::GPU_SCHEDULER->current_time(),
//                       index,
//                       p_index,
//                       pi->h_person_update_info()[index].person_latest_update_time,
//                       Model::GPU_SCHEDULER->current_time(),
//                       pi->h_person_update_info()[index].parasite_update_function_type[p_index],
//                       pi->h_person_update_info()[index].parasite_genotype[p_index],
//                       pi->h_person_update_info()[index].parasite_last_update_log10_parasite_density[p_index]);
//            }
//        }
//    }
//
//    ThrustTVectorDevice<GPU::Person::PersonUpdateInfo> d_person_update_info(pi->h_person_update_info().size());
//    thrust::copy(pi->h_person_update_info().begin(),
//                 pi->h_person_update_info().end(),
//                 d_person_update_info.begin());

//    /*
//     * Filter out person with no clonal parasite
//     * */
//    size_t n_has_parasites = thrust::count_if(thrust::device,
//                                              d_person_update_info.begin(),
//                                              d_person_update_info.end(),
//                                              person_has_parasites());
//    printf("GPU::PopulationKernel::update_all_individuals n_has_parasites %d\n",n_has_parasites);
//    ThrustTVectorDevice<GPU::Person::PersonUpdateInfo> d_person_update_info_has_parasites(n_has_parasites);
//    thrust::copy_if(thrust::device,
//                    d_person_update_info.begin(),
//                    d_person_update_info.end(),
//                    d_person_update_infohas_parasites.begin(),
//                    person_has_parasites());
//    check_cuda_error(hipGetLastError());

    for(int index = 1040; index <= 1045; index++){
        if(pi->h_person_update_info()[index].parasite_size > 0){
            for(int p_index = 0; p_index < pi->h_person_update_info()[index].parasite_size; p_index++){
                printf("%d %d HOST update_all_individuals before update parasite %d %d %d %d %s %f\n",
                       Model::GPU_SCHEDULER->current_time(),
                       index,
                       p_index,
                       pi->h_person_update_info()[index].person_latest_update_time,
                       Model::GPU_SCHEDULER->current_time(),
                       pi->h_person_update_info()[index].parasite_update_function_type[p_index],
                       pi->h_person_update_info()[index].parasite_genotype[p_index],
                       pi->h_person_update_info()[index].parasite_last_update_log10_parasite_density[p_index]);
            }
        }
    }

    /*
     * Update clonal parasite density
     * Here we don't copy Model::CONFIG->parasite_density_level()
     * to device because there is no vector or pointers in this struct
     * If it has any vector or pointers, we need to copy it to device first.
     * In the same manner we will need to copy ImmuneSystemInformation to device.
     * */
    ImmuneSystemInformation *d_immune_system_information;
    hipMalloc((void**)&d_immune_system_information, sizeof(ImmuneSystemInformation));
    hipMemcpy(d_immune_system_information, &Model::CONFIG->immune_system_information(), sizeof(ImmuneSystemInformation), hipMemcpyHostToDevice);
    check_cuda_error(hipGetLastError());
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = (pi->h_person_update_info().size() + n_threads - 1)/n_threads;
    all_individuals_update_parasite_kernel<<<block_size,n_threads>>>(pi->h_person_update_info().size(),
                                                            Model::GPU_SCHEDULER->current_time(),
                                                            Model::CONFIG->parasite_density_level(),
                                                            d_immune_system_information,
                                                            thrust::raw_pointer_cast(d_person_update_info.data())
                                                            );
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    for(int index = 1040; index <= 1045; index++){
        if(pi->h_person_update_info()[index].drug_in_blood_size > 0){
            for(int d_index = 0; d_index < pi->h_person_update_info()[index].drug_in_blood_size; d_index++){
                if(pi->h_person_update_info()[index].drug_in_blood_type_id[d_index] != -1){
                    int d_type_id = pi->h_person_update_info()[index].drug_in_blood_type_id[d_index];
                    printf("%d %d HOST update_all_individuals before update drug %d %d %d %f %f\n",
                           Model::GPU_SCHEDULER->current_time(),
                           index,
                           pi->h_person_update_info()[index].drug_start_time[d_type_id],
                           pi->h_person_update_info()[index].drug_last_update_time[d_type_id],
                           d_type_id,
                           pi->h_person_update_info()[index].drug_starting_value[d_type_id],
                           pi->h_person_update_info()[index].drug_last_update_value[d_type_id]);
                }
            }
        }
    }

    block_size = (d_person_update_info.size() + n_threads - 1)/n_threads;
    all_individuals_update_drug_kernel<<<block_size,n_threads>>>(d_person_update_info.size(),
                                                                 Model::GPU_SCHEDULER->current_time(),
                                                                 Model::CONFIG->parasite_density_level(),
                                                                 d_immune_system_information,
                                                                 thrust::raw_pointer_cast(d_person_update_info.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    /*
     * Copy back to host
     * */

    thrust::copy(d_person_update_info.begin(),
                 d_person_update_info.end(),
                 pi->h_person_update_info().begin());

    for(int index = 1040; index <= 1045; index++){
        if(pi->h_person_update_info()[index].drug_in_blood_size > 0){
            for(int d_index = 0; d_index < pi->h_person_update_info()[index].drug_in_blood_size; d_index++){
                if(pi->h_person_update_info()[index].drug_in_blood_type_id[d_index] != -1){
                    int d_type_id = pi->h_person_update_info()[index].drug_in_blood_type_id[d_index];
                    printf("%d %d HOST update_all_individuals after update drug %d %d %d %f %f\n",
                           Model::GPU_SCHEDULER->current_time(),
                           index,
                           pi->h_person_update_info()[index].drug_start_time[d_type_id],
                           pi->h_person_update_info()[index].drug_last_update_time[d_type_id],
                           d_type_id,
                           pi->h_person_update_info()[index].drug_starting_value[d_type_id],
                           pi->h_person_update_info()[index].drug_last_update_value[d_type_id]);
                }
            }
        }
    }

    for(int index = 1040; index <= 1045; index++){
        if(pi->h_person_update_info()[index].parasite_size > 0){
            for(int p_index = 0; p_index < pi->h_person_update_info()[index].parasite_size; p_index++){
                printf("%d %d HOST update_all_individuals after update parasite %d %d %d %d %s %f\n",
                       Model::GPU_SCHEDULER->current_time(),
                       index,
                       p_index,
                       pi->h_person_update_info()[index].person_latest_update_time,
                       Model::GPU_SCHEDULER->current_time(),
                       pi->h_person_update_info()[index].parasite_update_function_type[p_index],
                       pi->h_person_update_info()[index].parasite_genotype[p_index],
                       pi->h_person_update_info()[index].parasite_last_update_log10_parasite_density[p_index]);
            }
        }
    }

    if(Model::GPU_SCHEDULER->current_time() > 8)
        exit(0);
}
