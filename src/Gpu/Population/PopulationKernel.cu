#include "hip/hip_runtime.h"
//
// Created by kient on 12/31/2023.
//

#include <hiprand/hiprand_kernel.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include "PopulationKernel.cuh"
#include "Population.cuh"
#include "Model.h"
#include "Core/Config/Config.h"
#include "Gpu/MDC/ModelDataCollector.cuh"
#include "Spatial/SpatialModel.hxx"
#include "Core/Random.h"
#include "Gpu/Utils/Utils.cuh"
#include "Gpu/Core/Random.cuh"
#include "Gpu/Population/Properties/PersonIndexByLocationMovingLevel.cuh"
#include "Properties/PersonIndexGPU.cuh"
#include "ClonalParasitePopulation.cuh"
#include "SingleHostClonalParasitePopulations.cuh"
#include "ImmuneSystem.cuh"

GPU::PopulationKernel::PopulationKernel() {
}

void GPU::PopulationKernel::init() {
    h_ie_foi_N_days_all_locations = TVector<double>(Model::CONFIG->number_of_locations()
            *Model::CONFIG->number_of_tracking_days());
}
/*
 * Calculate number of circulations from each location
 * this if from poisson distribution with mean = popsize_residence_by_location * circulation_percent
 * d_n_circulations_from_locations is number of circulations from each location
 * */
__global__ void calculate_circulation_number(int n_locations,
                                             int* d_popsize_residence_by_location,
                                             double circulation_percent,
                                             hiprandState* d_state,
                                             int* d_n_circulations_from_locations) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n_locations; index += stride) {
        if (d_popsize_residence_by_location[index] == 0 || circulation_percent == 0.0) return;
        auto poisson_means = d_popsize_residence_by_location[index] * circulation_percent;
        d_n_circulations_from_locations[index] = hiprand_poisson(&d_state[index], poisson_means);
    }
    d_state[thread_index] = local_state;
}

/*
 * Calculate relative out movement to destination locations
 * this is get_v_relative_out_movement_to_destination in CPU v4.0 version
 * Output is relative out movement from each location to each destination location
 * So 9 locations will have 9*9 = 81 values
 * */
__global__ void calculate_circulation_probabilities(int n_locations,
                                                    int* d_popsize_residence_by_location,
                                                    double* d_spatial_model_parameters,
                                                    double* d_spatial_model_travels,
                                                    int* d_district_vector,
                                                    double* d_distance_vector,
                                                    double* d_relative_outmovement,
                                                    int* d_from_indices,
                                                    int* d_target_indices) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_locations; index += stride) {
        int from_location = index / n_locations;
        int target_location = index % n_locations;
        if(d_distance_vector[from_location * n_locations + target_location] == 0.0) return;
        double distance = d_distance_vector[from_location * n_locations + target_location];
        double kernel = pow(1 + (distance / d_spatial_model_parameters[2]), (-d_spatial_model_parameters[1]));
        double probability = pow(d_popsize_residence_by_location[from_location], d_spatial_model_parameters[0]) * kernel;
        probability = probability / (1 + d_spatial_model_travels[from_location] + d_spatial_model_travels[target_location] );
        if (d_district_vector[from_location] == static_cast<int>(d_spatial_model_parameters[3]) &&
            d_district_vector[target_location] == static_cast<int>(d_spatial_model_parameters[3])) {
            probability /= d_spatial_model_parameters[4];
        }
        d_relative_outmovement[index] = probability;
        d_from_indices[index] = from_location;
        d_target_indices[index] = target_location;
    }
}

void GPU::PopulationKernel::calculate_circulate_locations(int n_locations,ThrustTVectorDevice<int> &d_n_circulations_from_locations,ThrustTVectorDevice<double> &d_relative_outmovement_from_target_,
                                         ThrustTVectorDevice<int> &d_all_location_from_indices,ThrustTVectorDevice<int> &d_all_location_target_indices) {
    //Has to get pointer to device otherwise it will copy vector from host to device
    d_ce_popsize_residence_by_location = Model::GPU_DATA_COLLECTOR->popsize_residence_by_location();
    d_ce_spatial_model_parameters = Model::CONFIG->spatial_model()->getSpatialModelParameters();
    d_ce_spatial_model_travels = Model::CONFIG->spatial_model()->getSpatialModelTravels();
    d_ce_spatial_districts = Model::CONFIG->h_spatial_districts;
    d_ce_spatial_distances = Model::CONFIG->h_spatial_distances;

    //All probabilities because thrust run all arrays at the same time. If use 1 array then values are overwritten.
    d_n_circulations_from_locations.resize(n_locations,0);

    //Get circulations by location
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = (d_n_circulations_from_locations.size() + n_threads - 1)/n_threads;
    calculate_circulation_number<<<block_size,n_threads>>>(n_locations,
                                                           thrust::raw_pointer_cast(d_ce_popsize_residence_by_location.data()),
                                                           Model::CONFIG->circulation_info().circulation_percent,
                                                           Model::GPU_RANDOM->d_states,
                                                           thrust::raw_pointer_cast(d_n_circulations_from_locations.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    //Get outmovement probabilities
    d_relative_outmovement_from_target_.resize(n_locations*n_locations,0.0);
    d_all_location_from_indices.resize(n_locations*n_locations,0);
    d_all_location_target_indices.resize(n_locations*n_locations,0);
    n_threads = Model::CONFIG->gpu_config().n_threads;
    block_size = (d_relative_outmovement_from_target_.size() + n_threads - 1)/n_threads;
    calculate_circulation_probabilities<<<block_size,n_threads>>>(Model::CONFIG->number_of_locations(),
                                                                  thrust::raw_pointer_cast(d_ce_popsize_residence_by_location.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_model_parameters.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_model_travels.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_districts.data()),
                                                                  thrust::raw_pointer_cast(d_ce_spatial_distances.data()),
                                                                  thrust::raw_pointer_cast(d_relative_outmovement_from_target_.data()),
                                                                  thrust::raw_pointer_cast(d_all_location_from_indices.data()),
                                                                  thrust::raw_pointer_cast(d_all_location_target_indices.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
}

/*
 * Calculate moving level density at each location, output size is n_location*n_moving_level
 * This value is for each destination locations
 * */
__global__ void calculate_moving_level_density_kernel(int n_locations,
                                               int n_moving_levels,
                                               thrust::tuple<int,int> *d_circulation_indices,
                                               int* d_popsize_by_moving_level,
                                               double* d_moving_level_value,
                                               double* d_moving_level_density) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_moving_levels; index += stride) {
        int circulate_loc_index = index / n_moving_levels;
        int from_location = thrust::get<0>(d_circulation_indices[circulate_loc_index]);
        int moving_level = index % n_moving_levels;
        if(d_popsize_by_moving_level[from_location*n_moving_levels+moving_level] == 0 || d_moving_level_value[moving_level] == 0.0) return;
        d_moving_level_density[index] = d_popsize_by_moving_level[from_location*n_moving_levels+moving_level] * d_moving_level_value[moving_level];
    }
}

void GPU::PopulationKernel::calculate_moving_level_density(ThrustTuple2VectorDevice<int,int> d_circulation_indices,ThrustTVectorDevice<double> &d_moving_level_density) {
    if(d_circulation_indices.size() == 0){
        return;
    }
    d_moving_level_density.resize(d_circulation_indices.size()*Model::CONFIG->circulation_info().number_of_moving_levels);
    d_ce_popsize_by_moving_level = Model::CONFIG->h_popsize_by_moving_level;
    d_ce_moving_level_value = Model::CONFIG->circulation_info().v_moving_level_value;
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = (d_moving_level_density.size() + n_threads - 1)/n_threads;
    calculate_moving_level_density_kernel<<<block_size,n_threads>>>(d_circulation_indices.size(),
                                                             Model::CONFIG->circulation_info().number_of_moving_levels,
                                                             thrust::raw_pointer_cast(d_circulation_indices.data()),
                                                             thrust::raw_pointer_cast(d_ce_popsize_by_moving_level.data()),
                                                             thrust::raw_pointer_cast(d_ce_moving_level_value.data()),
                                                             thrust::raw_pointer_cast(d_moving_level_density.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
}

template <typename T>
struct notZero : public thrust::unary_function<T,bool> {
    __host__ __device__
    bool operator()(T x)
    {
        return x != 0;
    }
};

struct copyNotZero : public thrust::unary_function<unsigned int,bool>{
    __host__ __device__
    bool operator()(unsigned int x) {
        return x != 0;
    }
};

template <typename T>
struct isOne : public thrust::unary_function<T,bool> {
    __host__ __device__
    bool operator()(T x)
    {
        return x == 1;
    }
};

struct circulateLess{
    __host__ __device__
    bool operator()(const thrust::tuple<int,int,int,unsigned int>& t1, thrust::tuple<int,int,int,unsigned int>& t2)
    {
        if(t1.get<3>() < t2.get<3>())
            return true;
        if(t1.get<3>() > t2.get<3>())
            return false;
        return t1.get<3>() < t2.get<3>();
    }
};

/*
 * Get 3 vectors of from_location, target_location and moving_level for removing zero from
 * d_circulation_indices and d_n_circulations_all_loc_ml
 * */
__global__ void extract_locations_and_moving_levels(int n_locations,
                                          int n_moving_levels,
                                          thrust::tuple<int,int> *d_circulation_indices,
                                          unsigned int *d_n_circulations_all_loc_ml,
                                          int* d_from_indices,
                                          int* d_target_indices,
                                          int* d_moving_levels){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < n_locations*n_moving_levels; index += stride) {
        if(d_n_circulations_all_loc_ml[index] == 0) return;
        int circulate_index = index / n_moving_levels;
        d_from_indices[index] = thrust::get<0>(d_circulation_indices[circulate_index]);
        d_target_indices[index] = thrust::get<1>(d_circulation_indices[circulate_index]);
        d_moving_levels[index] = index % n_moving_levels;
    }
}

/*
 * Combine 2 vectors to 1 vectors, for sorting
 * */
__global__ void zip_location_indices_and_n_circulations(int size,
                                          thrust::tuple<int,int,int>* d_circulations_all_loc_ml_indices_no_zero,
                                          unsigned int* d_n_circulations_all_loc_ml_no_zero,
                                          thrust::tuple<int,int,int,unsigned int>* d_circulate_all_loc_ml_today){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
        if(d_n_circulations_all_loc_ml_no_zero[index] == 0) return;
        d_circulate_all_loc_ml_today[index] = thrust::make_tuple(thrust::get<0>(d_circulations_all_loc_ml_indices_no_zero[index]),
                                                                     thrust::get<1>(d_circulations_all_loc_ml_indices_no_zero[index]),
                                                                     thrust::get<2>(d_circulations_all_loc_ml_indices_no_zero[index]),
                                                                     d_n_circulations_all_loc_ml_no_zero[index]);
    }
}

/*
 * Get random person index at each moving level in each location in all locations
 * Note that this is just filling 1 person index for each moving level in each location
 * so n_persons data in d_circulate_all_loc_ml_today is not used but will be embedded in d_circulate_person_indices_today
 * */
__global__ void fill_circulate_person_indices(int n_locations,
                                             int n_moving_levels,
                                             hiprandState *d_state,
                                             thrust::tuple<int,int,int,unsigned int> *d_circulate_all_loc_ml_today,
                                             int * d_popsize_by_loc_ml,
                                             thrust::tuple<int,int,int,unsigned int,int> *d_circulate_person_indices_today){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState local_state = d_state[thread_index];
    for(int index = thread_index; index < n_locations; index += stride) {
        if(thrust::get<3>(d_circulate_all_loc_ml_today[index]) == 0) return;
        int from_location = thrust::get<0>(d_circulate_all_loc_ml_today[index]);
        int target_location = thrust::get<1>(d_circulate_all_loc_ml_today[index]);
        int moving_level = thrust::get<2>(d_circulate_all_loc_ml_today[index]);
        unsigned int n_persons = thrust::get<3>(d_circulate_all_loc_ml_today[index]);
        int size = d_popsize_by_loc_ml[from_location*n_moving_levels+moving_level];
        if(size == 0 || n_persons == 0) return;
        /*
         * Random uniform to get person index
         * To get result same as gsl, using casting method, which is [0,n-1]
         * ceiling methos is [1,n]
         * https://github.com/nglee/so_answers/blob/master/cuda/170426/kernel.cu
         * */
//        printf("kernel %d from_location: %d, moving_level: %d, popsize_loc_ml: %d size: %d\n",index,
//               from_location,moving_level,popsize_loc_ml,d_n_circulations_all_loc_ml_no_zero[index]);
        if(n_persons == 1){
            int p_index = hiprand_uniform(&local_state) * size;
            d_circulate_person_indices_today[index] = thrust::make_tuple(from_location,target_location,moving_level,n_persons,p_index);
//        printf("kernel %d from_location: %d, moving_level: %d, popsize_loc_ml: %d, size: %d, p_index: %d\n",index,from_location,moving_level,
//               popsize_loc_ml,d_n_circulations_all_loc_ml_no_zero[index],p_index);
        }
        else{
            d_circulate_person_indices_today[index] = thrust::make_tuple(from_location,target_location,moving_level,n_persons,-1);
        }
    }
    d_state[thread_index] = local_state;
}

/*
 * To speed up circulation process using GPU, doing following steps:
 * 1. Calculate number of circulations and leavers, same as CPU
 * 2. Filter out location with zero circulation
 * 3. Parallel multinomial in all non-zero circulation locations,
 * each location select number of leavers in each moving level
 * 4. Sort the result of multinomial, so all location and moving level with 1 person will be filled first on GPU
 * 5. For the rest, fill on CPU
 * */
void GPU::PopulationKernel::perform_circulation_event() {
    auto tp_start = std::chrono::high_resolution_clock::now();

    /*
     * Calculate probability of leaving location in all locations (n_location*n_location)
     * Also get indices from and to arrays
     * */
    Model::GPU_POPULATION_KERNEL->calculate_circulate_locations(Model::CONFIG->number_of_locations(),
                                                         d_ce_n_circulations_from_locations,
                                                         d_ce_relative_outmovement_from_target,
                                                         d_ce_all_location_from_indices, d_ce_all_location_target_indices);

    /*
     * Calculate number of leavers in all locations
     * */
    ThrustTVectorDevice<unsigned int> d_num_leavers_from_target_(d_ce_relative_outmovement_from_target.size(), 0);
    Model::GPU_RANDOM->random_multinomial(Model::CONFIG->number_of_locations(),
                                          Model::CONFIG->number_of_locations(),
                                          d_ce_n_circulations_from_locations,
                                          d_ce_relative_outmovement_from_target,
                                          d_num_leavers_from_target_);
    size_t no_zero_size = thrust::count_if(d_num_leavers_from_target_.begin(), d_num_leavers_from_target_.end(), notZero<unsigned int>());
    /*
     * Remove zero values in d_num_leavers_from_target_ and d_n_circulations_from_locations
     * d_circulations_indices_no_zero index is not location index, its index is index of n_locations*n_locations
     * thrust::get<0>(d_circulations_indices_no_zero[i]) is from location index
     * thrust::get<1>(d_circulations_indices_no_zero[i]) is to location index
     * */
    ThrustTuple2VectorDevice<int,int> d_circulations_indices_no_zero(no_zero_size);

    /*
     * Remove zero values in d_num_leavers_from_target_ and d_n_circulations_from_locations
     * scan d_num_leavers_from_target_ and copy non-zero from & to locations to d_circulations_indices_no_zero
     * This is to reduce compute time
     */
    auto loc_index_begin = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.begin(), d_ce_all_location_target_indices.begin()));
    auto loc_index_end = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.end(), d_ce_all_location_target_indices.end()));
    thrust::copy_if(thrust::device,
                     loc_index_begin,
                     loc_index_end,
                     d_num_leavers_from_target_.begin(),
                     d_circulations_indices_no_zero.begin(),
                     copyNotZero());
    thrust::device_vector<unsigned int>::iterator nend = thrust::remove(thrust::device,d_num_leavers_from_target_.begin(),d_num_leavers_from_target_.end(),0);
    ThrustTVectorDevice<unsigned int> d_num_leavers_from_target_no_zero(no_zero_size);
    thrust::copy(d_num_leavers_from_target_.begin(),nend,d_num_leavers_from_target_no_zero.begin());

    /*
     * d_num_leavers_from_target_no_zero is non-zero leavers in all locations
     * */
    int total_leavers = thrust::reduce(thrust::device,d_num_leavers_from_target_no_zero.begin(),d_num_leavers_from_target_no_zero.end());
//    LOG_IF(total_leavers == 0, DEBUG) << "[Population] Update population circulation GPU total_leavers = 0";
    if(total_leavers == 0) return;

    /*
     * Calculate moving level density at each location, output size is n_location*n_moving_level
     * n_location is no-zero indices
     */
    ThrustTVectorDevice<double> d_moving_level_density;
    Model::GPU_POPULATION_KERNEL->calculate_moving_level_density(d_circulations_indices_no_zero,d_moving_level_density);

    ThrustTVectorDevice<unsigned int> d_n_circulations_all_loc_ml(d_moving_level_density.size(),0);
    Model::GPU_RANDOM->random_multinomial(d_circulations_indices_no_zero.size(),
                                         Model::CONFIG->circulation_info().number_of_moving_levels,
                                         d_num_leavers_from_target_no_zero,
                                         d_moving_level_density,
                                         d_n_circulations_all_loc_ml);
    /*
     * d_circulations_indices_no_zero and d_n_circulations_all_loc_ml not the same size
     */
//    TVector<unsigned int> h_n_circulations_all_loc_ml = d_n_circulations_all_loc_ml;
//    ThrustT2TupleVector<int,int> h_circulations_indices_no_zero = d_circulations_indices_no_zero;
//    for(int i = 0; i < h_n_circulations_all_loc_ml.size(); i++){
//        int circulate_loc_index = i / Model::CONFIG->circulation_info().number_of_moving_levels;
//        int from_location = thrust::get<0>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int target_location = thrust::get<1>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int moving_level = i % Model::CONFIG->circulation_info().number_of_moving_levels;
//        if(h_n_circulations_all_loc_ml[i] == 0) continue;
//        printf("%d from %d to %d moving level %d size %d\n",i,from_location,target_location,moving_level,
//               h_n_circulations_all_loc_ml[i]);
//    }
//    printf("\n");

    /*
     * Remove zero values in d_n_circulations_all_loc_ml
     * First, extract from and to locations from d_n_circulations_all_loc_ml to 2 vectors
     */
    d_ce_all_location_from_indices.resize(d_n_circulations_all_loc_ml.size());
    d_ce_all_location_target_indices.resize(d_n_circulations_all_loc_ml.size());
    d_ce_all_moving_levels.resize(d_n_circulations_all_loc_ml.size());
    if(d_n_circulations_all_loc_ml.size() == 0) return;
    int n_threads = Model::CONFIG->gpu_config().n_threads;
    int block_size = (d_n_circulations_all_loc_ml.size() + n_threads - 1)/n_threads;
    extract_locations_and_moving_levels<<<block_size,n_threads>>>(d_circulations_indices_no_zero.size(),
                                                      Model::CONFIG->circulation_info().number_of_moving_levels,
                                                      thrust::raw_pointer_cast(d_circulations_indices_no_zero.data()),
                                                      thrust::raw_pointer_cast(d_n_circulations_all_loc_ml.data()),
                                                      thrust::raw_pointer_cast(d_ce_all_location_from_indices.data()),
                                                      thrust::raw_pointer_cast(d_ce_all_location_target_indices.data()),
                                                      thrust::raw_pointer_cast(d_ce_all_moving_levels.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());
//    TVector<int> h_all_location_from_indices = d_all_location_from_indices;
//    TVector<int> h_all_location_target_indices = d_all_location_target_indices;
//    TVector<int> h_all_moving_levels = d_all_moving_levels;
//    for(int i = 0; i < h_n_circulations_all_loc_ml.size(); i++){
//        int circulate_loc_index = i / Model::CONFIG->circulation_info().number_of_moving_levels;
//        int from_location = thrust::get<0>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int target_location = thrust::get<1>(h_circulations_indices_no_zero[circulate_loc_index]);
//        int moving_level = i % Model::CONFIG->circulation_info().number_of_moving_levels;
//        int from_location2 = h_all_location_from_indices[i];
//        int target_location2 = h_all_location_target_indices[i];
//        int moving_level2 = h_all_moving_levels[i];
//        if(h_n_circulations_all_loc_ml[i] == 0) continue;
//        printf("%d from %d to %d moving level %d (%d to %d moving level %d) size %d\n",i,from_location,target_location,moving_level,
//               from_location2,target_location2,moving_level2,h_n_circulations_all_loc_ml[i]);
//    }
//    printf("\n");
    /*
     * Remove zero values in d_n_circulations_all_loc_ml
     * scan d_n_circulations_all_loc_ml and copy non-zero from & to locations from 2 vectors
     * to d_circulations_all_loc_ml_indices_no_zero
     */
    no_zero_size = thrust::count_if(d_n_circulations_all_loc_ml.begin(), d_n_circulations_all_loc_ml.end(), notZero<unsigned int>());
    auto loc_index_begin_2 = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.begin(),
                                                                          d_ce_all_location_target_indices.begin(),
                                                                          d_ce_all_moving_levels.begin()));
    auto loc_index_end_2 = thrust::make_zip_iterator(thrust::make_tuple(d_ce_all_location_from_indices.end(),
                                                                        d_ce_all_location_target_indices.end(),
                                                                        d_ce_all_moving_levels.end()));
    ThrustTuple3VectorDevice<int,int,int> d_circulations_all_loc_ml_indices_no_zero(no_zero_size);
    thrust::copy_if(thrust::device,
                    loc_index_begin_2,
                    loc_index_end_2,
                    d_n_circulations_all_loc_ml.begin(),
                    d_circulations_all_loc_ml_indices_no_zero.begin(),
                    copyNotZero());
    nend = thrust::remove(thrust::device,d_n_circulations_all_loc_ml.begin(),d_n_circulations_all_loc_ml.end(),0);
    ThrustTVectorDevice<unsigned int> d_n_circulations_all_loc_ml_no_zero(no_zero_size);
    thrust::copy(d_n_circulations_all_loc_ml.begin(),nend,d_n_circulations_all_loc_ml_no_zero.begin());

    /*
     * d_circulations_all_loc_ml_indices_no_zero is tuple of from_location, target_location and moving level
     * d_n_circulations_all_loc_ml_no_zero is non-zero circulation number in all locations and moving levels
     * d_n_circulations_all_loc_ml_no_zero and d_circulations_all_loc_ml_indices_no_zero are the same size and order
     * d_circulate_all_loc_ml_today is tuple of from_location, target_location, moving_level, person_index
     * */
    int total_circulations = thrust::reduce(thrust::device,d_n_circulations_all_loc_ml_no_zero.begin(),d_n_circulations_all_loc_ml_no_zero.end());
    ThrustTuple4VectorDevice<int,int,int,unsigned int> d_circulate_all_loc_ml_today(d_circulations_all_loc_ml_indices_no_zero.size(),thrust::make_tuple(-1,-1,-1,0));
    block_size = (d_circulate_all_loc_ml_today.size() + n_threads - 1)/n_threads;
    zip_location_indices_and_n_circulations<<<block_size,n_threads>>>(d_circulate_all_loc_ml_today.size(),
                                                                      thrust::raw_pointer_cast(d_circulations_all_loc_ml_indices_no_zero.data()),
                                                                      thrust::raw_pointer_cast(d_n_circulations_all_loc_ml_no_zero.data()),
                                                                      thrust::raw_pointer_cast(d_circulate_all_loc_ml_today.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    thrust::sort(thrust::device,d_circulate_all_loc_ml_today.begin(),d_circulate_all_loc_ml_today.end(),circulateLess());

//    ThrustT4TupleVector<int,int,int,unsigned int> h_circulate_all_loc_ml_today = d_circulate_all_loc_ml_today;
//    for(int i = 0; i < h_circulate_all_loc_ml_today.size(); i++){
//        int from_location = thrust::get<0>(h_circulate_all_loc_ml_today[i]);
//        int target_location = thrust::get<1>(h_circulate_all_loc_ml_today[i]);
//        int moving_level = thrust::get<2>(h_circulate_all_loc_ml_today[i]);
//        unsigned int size = thrust::get<3>(h_circulate_all_loc_ml_today[i]);
//        printf("%d from %d to %d moving level %d size %d\n",i,from_location,target_location,moving_level,size);
//    }
//    printf("\n");

    /*
     * Random persons based on d_n_circulations_all_loc_ml_no_zero
     * This needs to be done in 2 passes, 1st pass to fill all index with 1 person first, 1st pass is done on GPU
     * 2nd pass to fill all index with n_persons > 1, 2nd pass is done on CPU
     * In this is the first pass, fill all index in d_circulate_person_indices_today
     * with 1 person first
     * */
    ThrustTuple5VectorDevice<int,int,int,unsigned int,int> d_circulate_person_indices_today(total_circulations,thrust::make_tuple(-1,-1,-1,-1,-1));
    block_size = (d_n_circulations_all_loc_ml_no_zero.size() + n_threads - 1)/n_threads;
    fill_circulate_person_indices<<<block_size,n_threads>>>(d_circulate_all_loc_ml_today.size(),
                                                           Model::CONFIG->circulation_info().number_of_moving_levels,
                                                           Model::GPU_RANDOM->d_states,
                                                           thrust::raw_pointer_cast(d_circulate_all_loc_ml_today.data()),
                                                           thrust::raw_pointer_cast(d_ce_popsize_by_moving_level.data()),
                                                           thrust::raw_pointer_cast(d_circulate_person_indices_today.data()));
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());

    /*
     * Check if there is any index with n_persons > 1 and random on CPU
     * otherwise schedule person events
     * */
    ThrustTuple5VectorHost<int,int,int,unsigned int,int> h_circulate_person_indices_today = d_circulate_person_indices_today;
    auto *pi = Model::GPU_POPULATION->get_person_index<GPU::PersonIndexByLocationMovingLevel>();
    for(int i = 0; i < d_circulate_all_loc_ml_today.size();i++){
        int from_location = h_circulate_person_indices_today[i].get<0>();
        int target_location = h_circulate_person_indices_today[i].get<1>();
        int moving_level = h_circulate_person_indices_today[i].get<2>();
        int n_persons = h_circulate_person_indices_today[i].get<3>();
        auto size = static_cast<int>(pi->vPerson()[from_location][moving_level].size());
        if (size==0) continue;
        if(n_persons == 1){
            int p_index = h_circulate_person_indices_today[i].get<4>();
            GPU::Person* p = pi->vPerson()[from_location][moving_level][p_index];
            assert(p->host_state()!=Person::DEAD);
            p->today_target_locations()->push_back(target_location);
            p->randomly_choose_target_location();
//            printf("i %d GPU from %d to %d moving level %d n_persons %d p_index %d\n",
//                   i,
//                   from_location,target_location,moving_level,
//                   n_persons,p_index);
        }
        else{
            for(int j = 0; j < n_persons; j++) {
                int p_index = Model::RANDOM->random_uniform(size);
                GPU::Person* p = pi->vPerson()[from_location][moving_level][p_index];
                assert(p->host_state()!=Person::DEAD);
                p->today_target_locations()->push_back(target_location);
                p->randomly_choose_target_location();
//                printf("i %d j %d CPU from %d to %d moving level %d n_persons %d p_index %d\n",
//                       i,j,
//                       from_location,target_location,moving_level,
//                       n_persons,p_index);
            }
        }
    }

    if(Model::CONFIG->debug_config().enable_debug_text){
        auto lapse = std::chrono::high_resolution_clock::now() - tp_start;
        LOG_IF(Model::GPU_SCHEDULER->current_time() % Model::CONFIG->debug_config().log_interval == 0, INFO)
        << "[GPU Population] Update population circulation GPU (" << d_circulations_indices_no_zero.size() << " " << d_num_leavers_from_target_no_zero.size()
        << " " << total_leavers << " " << total_circulations << ") event time: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(lapse).count() << " ms ";
    }
}

void GPU::PopulationKernel::calculate_n_person_bitten_today(int n_locations,
                                                      ThrustTVectorDevice<double> &d_foi_all_locations,
                                                      ThrustTVectorDevice<int> &d_n_person_bitten_today_all_locations){


}

void GPU::PopulationKernel::perform_infection_event() {
    auto tp_start = std::chrono::high_resolution_clock::now();
    auto tracking_index = Model::GPU_SCHEDULER->current_time() % Model::CONFIG->number_of_tracking_days();

    /*
     * Calculate probability of leaving location in all locations (n_location*n_location)
     * Also get indices from and to arrays
     * */

    ThrustTVectorDevice<double> d_foi_all_locations;
    ThrustTVectorDevice<int> d_n_person_bitten_today_all_locations;
    calculate_n_person_bitten_today(Model::CONFIG->number_of_locations(),
                                    d_foi_all_locations,d_n_person_bitten_today_all_locations);
}

__global__ void update_current_foi_kernel(int size){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
    }
}

void GPU::PopulationKernel::update_current_foi(){

}

/*
 * In order to use virtual in kernel, the base class needs to be instanced on GPU
 * */
__global__ void set_gpu_update_function_kernel(int size,
                                               GPU::ParasiteDensityUpdateFunction** d_function,
//                                               GPU::ImmuneSystem** d_immune_system,
                                               int* type){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
        switch(type[index]) {
            case 1:
                d_function[index] = new GPU::ClinicalUpdateFunction();
                break;
            case 2:
                d_function[index] = new GPU::ImmunityClearanceUpdateFunction();
                break;
            default:
                printf("[GPU::ClonalParasitePopulation] ERROR: GPU::ParasiteDensityUpdateFunction not set\n");
                break;
        }
//        d_immune_system[index] = new GPU::ImmuneSystem();
    }
}

/*
 * Set update function inside kernel
 * To do this the base class of update function needs to be on GPU first
 * Remember to run
 * h_cpp->set_h_parasite_density_level(Model::CONFIG->parasite_density_level());
 * h_cpp->allocate_on_gpu();
 * before calling this function
 * */
//__global__ void update_all_individuals_kernel(int size,
//                                              int current_time,
//                                              ParasiteDensityLevel h_parasite_density_level,
//                                              ImmuneSystemInformation h_immune_system_information,
////                                              GPU::ParasiteDensityUpdateFunction** d_update_function,
////                                              GPU::ImmuneSystem** d_immune_system,
//                                              GPU::ClonalParasitePopulation** d_cpp){
//    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
//    int stride = blockDim.x * gridDim.x;
//    for(int index = thread_index; index < size; index += stride) {
////        printf("index %d cpp %d\n",index,d_update_function[index]->type());
//        d_cpp[index]->update_gpu(h_parasite_density_level,h_immune_system_information,
////                          d_update_function[index],d_immune_system[index],
//                          current_time);
//    }
//}


/*
 * Set update function inside kernel
 * To do this the base class of update function needs to be on GPU first
 * Remember to run
 * h_cpp->set_h_parasite_density_level(Model::CONFIG->parasite_density_level());
 * h_cpp->allocate_on_gpu();
 * before calling this function
 * */
__global__ void update_all_individuals_kernel2(int size,
                                              int* d_person_id,
                                              int current_time,
                                              int* lastest_updated_time,
                                              ParasiteDensityLevel h_parasite_density_level,
                                              ImmuneSystemInformation *h_immune_system_information,
                                              GPU::ClonalParasitePopulation** d_cpp,
                                              GPU::Genotype** d_genotype,
                                              GPU::ParasiteDensityUpdateFunction** d_update_function,
                                              GPU::ImmuneSystem** d_immune_system){
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int index = thread_index; index < size; index += stride) {
        if (lastest_updated_time[index] == current_time) return;
        d_person_id[index],d_cpp[index]->get_current_parasite_density_gpu(d_update_function[index],
                                                                          d_genotype[index],
                                                                          d_immune_system[index],
                                                                          h_parasite_density_level,
                                                                          h_immune_system_information,
                                                                          current_time,
                                                                          lastest_updated_time[index]);
//        printf("kernel d_cpp %d %f\n",d_person_id[index],d_cpp[index]->get_current_parasite_density_gpu(d_update_function[index],
//                                                                                                        d_genotype[index],
//                                                                                                        d_immune_system[index],
//                                                                                                        h_parasite_density_level,
//                                                                                                        h_immune_system_information,
//                                                                                                        current_time,
//                                                                                                        lastest_updated_time[index]));
        lastest_updated_time[index] = current_time;
    }
}

void GPU::PopulationKernel::update_all_individuals(){
    auto *pi = Model::GPU_POPULATION->get_person_index<GPU::PersonIndexGPU>();
    TVector<GPU::ClonalParasitePopulation*> h_cpp;
    TVector<GPU::Genotype*> h_genotype;
    TVector<GPU::ImmuneSystem*> h_immune_system;
    TVector<int> h_cpp_update_function_type;
    TVector<int> h_person_id;
    TVector<int> h_latest_update_time;
    for(int i = 0; i < pi->h_persons().size(); i++){
        auto sh = pi->h_persons()[i]->all_clonal_parasite_populations();
        if(sh->size() == 0) continue;
        for(int j = 0; j < sh->size(); j++){
            if(sh->parasites()->at(j)->update_function() == nullptr) continue;
            if(sh->parasites()->at(j)->update_function()->type() == 0) continue;
            h_genotype.push_back(sh->parasites()->at(j)->genotype());
            h_immune_system.push_back(pi->h_persons()[i]->immune_system());
            h_cpp.push_back(sh->parasites()->at(j));
            h_cpp_update_function_type.push_back(sh->parasites()->at(j)->update_function()->type());
            h_person_id.push_back(pi->h_persons()[i]->id());
            h_latest_update_time.push_back(pi->h_persons()[i]->latest_update_time());
        }
    }
    if(h_cpp.size() == 0) return;

    printf("h_cpp size %d\n",h_cpp.size());
    const int h_cpp_size = h_cpp.size();

//    printf("Host:\n");
//    for (int i = 0; i < h_cpp_size; i++) {
//        std::cout << h_genotype[i]->test2() << std::endl;
//        std::cout << h_immune_system[i]->test_ << std::endl;
//        printf("%d type %d\n",i,h_cpp_update_function_type[i]);
//    }

    /*
     * Since ParasiteDensityUpdateFunction and ImmuneSystem are virtual classes,
     * they need to be instanced on GPU first
     * */
    int *d_cpp_update_function_type;
    hipMalloc((void**)&d_cpp_update_function_type, sizeof(int) * h_cpp_size);
    hipMemcpy(d_cpp_update_function_type, h_cpp_update_function_type.data(), sizeof(int) * h_cpp_size, hipMemcpyHostToDevice);
    check_cuda_error(hipGetLastError());

    GPU::ParasiteDensityUpdateFunction **d_update_function;
    hipMalloc((void**)&d_update_function, h_cpp_size*sizeof(GPU::ParasiteDensityUpdateFunction*));
    check_cuda_error(hipGetLastError());

//    GPU::ImmuneSystem **d_immune_system;
//    hipMalloc((void**)&d_immune_system, h_cpp_size*sizeof(GPU::ImmuneSystem*));
//    check_cuda_error(hipGetLastError());

    int n_threads = 256;
    int block_size = (h_cpp_size + n_threads - 1)/n_threads;
    set_gpu_update_function_kernel<<<block_size,n_threads>>>(h_cpp_size,
                                            d_update_function,
//                                            d_immune_system,
                                            d_cpp_update_function_type);
    hipDeviceSynchronize();
    check_cuda_error(hipGetLastError());


//    /*
//     * ClonalParasitePopulation and Genotype are not virtual classes,
//     * they can be copied from CPU
//     * refer to https://stackoverflow.com/questions/38978297/pointer-to-array-of-pointers-to-objects-in-cuda
//     * */
//    GPU::ClonalParasitePopulation** h_d_cpp = (GPU::ClonalParasitePopulation**)malloc(sizeof(GPU::ClonalParasitePopulation*) * h_cpp_size);
//    for (int i = 0; i < h_cpp_size; i++) {
//        // Allocate space for an Obj and assign
//        hipMalloc((void**)&h_d_cpp[i], sizeof(GPU::ClonalParasitePopulation));
//        // Copy the object to the device (only has single scalar field to keep it simple)
//        hipMemcpy(h_d_cpp[i], &(h_cpp[i]), sizeof(GPU::ClonalParasitePopulation), hipMemcpyHostToDevice);
//        check_cuda_error(hipGetLastError());
//    }
//
//    // Create a pointer which will point to device memory
//    GPU::ClonalParasitePopulation** d_d_cpp = NULL;
//    // Allocate space for 3 pointers on device at above location
//    hipMalloc((void**)&d_d_cpp, sizeof(GPU::ClonalParasitePopulation*) * h_cpp_size);
//    // Copy the pointers from the host memory to the device array
//    hipMemcpy(d_d_cpp, h_d_cpp, sizeof(GPU::ClonalParasitePopulation*) * h_cpp_size, hipMemcpyHostToDevice);
//    check_cuda_error(hipGetLastError());
//
//    GPU::Genotype** h_d_genotype = (GPU::Genotype**)malloc(sizeof(GPU::Genotype*) * h_cpp_size);
//    for (int i = 0; i < h_cpp_size; i++) {
//        // Allocate space for an Obj and assign
//        hipMalloc((void**)&h_d_genotype[i], sizeof(GPU::Genotype));
//        // Copy the object to the device (only has single scalar field to keep it simple)
//        hipMemcpy(h_d_genotype[i], &(h_genotype[i]), sizeof(GPU::Genotype), hipMemcpyHostToDevice);
//        check_cuda_error(hipGetLastError());
//    }
//
//    // Create a pointer which will point to device memory
//    GPU::Genotype** d_d_genotype = NULL;
//    // Allocate space for 3 pointers on device at above location
//    hipMalloc((void**)&d_d_genotype, sizeof(GPU::Genotype*) * h_cpp_size);
//    // Copy the pointers from the host memory to the device array
//    hipMemcpy(d_d_genotype, h_d_genotype, sizeof(GPU::Genotype*) * h_cpp_size, hipMemcpyHostToDevice);
//    check_cuda_error(hipGetLastError());
//
//    ImmuneSystemInformation *d_immune_system_information;
//    hipMalloc((void**)&d_immune_system_information, sizeof(ImmuneSystemInformation));
//    hipMemcpy(d_immune_system_information, &Model::CONFIG->immune_system_information(), sizeof(ImmuneSystemInformation), hipMemcpyHostToDevice);
//    check_cuda_error(hipGetLastError());
//
//    int *d_person_id;
//    hipMalloc((void**)&d_person_id, sizeof(int) * h_cpp_size);
//    hipMemcpy(d_person_id, h_person_id.data(), sizeof(int) * h_cpp_size, hipMemcpyHostToDevice);
//    check_cuda_error(hipGetLastError());
//
//    int *d_latest_update_time;
//    hipMalloc((void**)&d_latest_update_time, sizeof(int) * h_cpp_size);
//    hipMemcpy(d_latest_update_time, h_latest_update_time.data(), sizeof(int) * h_cpp_size, hipMemcpyHostToDevice);
//    check_cuda_error(hipGetLastError());
//
////    update_all_individuals_kernel2<<<block_size,n_threads>>>(h_cpp_size,
////                                                            d_person_id,
////                                                            Model::GPU_SCHEDULER->current_time(),
////                                                            d_latest_update_time,
////                                                            Model::CONFIG->parasite_density_level(),
////                                                            d_immune_system_information,
////                                                            d_d_cpp,
////                                                            d_d_genotype,
////                                                            d_update_function,
////                                                            d_immune_system);
////    hipDeviceSynchronize();
////    check_cuda_error(hipGetLastError());
//
////    for (int i = 0; i < h_cpp_size; i++) {
////        hipMemcpy(h_cpp[i], h_d_cpp[i], sizeof(GPU::ClonalParasitePopulation), hipMemcpyDeviceToHost);
//////        hipMemcpy(h_genotype[i], h_d_genotype[i], sizeof(GPU::Genotype), hipMemcpyDeviceToHost);
////        hipMemcpy(h_immune_system[i], h_d_immune_system[i], sizeof(GPU::ImmuneSystem), hipMemcpyDeviceToHost);
////        check_cuda_error(hipGetLastError());
////    }
//
//    // Write out
////    printf("D2H Host:\n");
////    for (int i = 0; i < h_cpp_size; i++) {
////        std::cout << h_cpp[i]->test2() << std::endl;
////        std::cout << h_genotype[i]->test_ << std::endl;
////    }
//
//    for(int i = 0; i < h_cpp_size; i++){
//        hipFree(h_d_cpp[i]);
//        hipFree(h_d_genotype[i]);
//    }
//    hipFree(d_immune_system);
//    hipFree(d_update_function);
//    hipFree(d_immune_system_information);
//    hipFree(d_person_id);
//    hipFree(d_latest_update_time);
//    h_cpp.clear();
//    h_genotype.clear();
//    h_immune_system.clear();
}
